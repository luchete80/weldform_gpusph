#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

/*! \file
 * Template implementation of the NeibsEngine in CUDA
 */

#include <stdexcept>

#include <stdio.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>

#include "define_buffers.h"
#include "engine_neibs.h"
#include "utils.h"

#include "textures.cuh"

#include "buildneibs_params.h"
#include "buildneibs_kernel.cu"

#include "vector_math.h"


/// Neighbor engine class
/*!	CUDANeibsEngine is an implementation of the abstract class AbstractNeibsEngine
 *	and is providing :
 *		- device constants upload to the device
 *		- device variables upload/download to/from the device
 *		- launch of sorting and reordering kernels
 *		- launch of neighbor list construction kernels
 *
 *	It is templatizd by:
 *	\tparam boundarytype : type of boundary
 *	\tparam periodicbound : type of periodic boundaries (0 ... 7)
 *	\tparam neibcount : true if we want to compute actual neighbors number

 *	\ingroup neibs
*/
template<SPHFormulation sph_formulation, /*typename ViscSpec, */BoundaryType boundarytype, Periodicity periodicbound, bool neibcount>
class CUDANeibsEngine : public AbstractNeibsEngine
{
public:

/** \name Constants upload/download and timing related function
 *  @{ */

/// Upload constants on the device
/*! This function upload neighbors search related constants on the device.
 * 	\param[in] simparams : pointer to simulation parameters structure
 * 	\param[in] physparams : pointer to physical parameters structure
 * 	\param[in] worldOrigin : origin of the simulation domain
 * 	\param[in] gridSize : size of computational domain in grid cells
 * 	\param[in] cellSize : size of each cell
 * 	\param[in] allocatedParticles : number of allocated particles
 */
void
setconstants(	const SimParams *simparams,		// pointer to simulation parameters structure (in)
				const PhysParams *physparams,		// pointer to physical parameters structure (in)
				float3 const& worldOrigin,			// origin of the simulation domain (in)
				uint3 const& gridSize,				// size of computational domain in grid cells (in)
				float3 const& cellSize,				// size of each cell (in)
				idx_t const& allocatedParticles)	// number of allocated particles (in)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neibboundpos), &simparams->neibboundpos, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblistsize), &simparams->neiblistsize, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));
}

/// Download maximum number of neighbors
/*! Download from device the maximum number of neighbors per particle
 *  computed by buildNeibsDevice kernel.
 *  \param[in] simparams : pointer to simulation parameters structure
 *  \param[in] physparams : pointer to physical parameters structure
 */
void
getconstants(	SimParams *simparams,	// pointer to simulation parameters structure (in)
				PhysParams *physparams)	// pointer to physical parameters structure (in)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&simparams->neibboundpos, HIP_SYMBOL(cuneibs::d_neibboundpos), sizeof(uint), 0));
}


/// Reset number of neighbors and interaction
/*! Reset number of neighbors and number of interactions stored
 * 	into GPU constant memory.
 */
void
resetinfo(void)
{
	int temp = 0;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_numInteractions), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxFluidBoundaryNeibs), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxVertexNeibs), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_hasMaxNeibs), &temp, sizeof(int)));
	temp = -1;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_hasTooManyNeibs), &temp, sizeof(int)));
}


/// Download number of neighbors and interactions
/*!	Download from GPU the maximum number of neighbors along with the
 * 	total number of interactions. Those data will be used to update a
 * 	TimingInfo structure.
 * 	\param[in, out] timingInfo : timing info struct where number of interactions and max
 * 	neighbors number will be updated
 */
void
getinfo(TimingInfo & timingInfo)	// timing info (in, out)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.numInteractions, HIP_SYMBOL(cuneibs::d_numInteractions), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.maxFluidBoundaryNeibs, HIP_SYMBOL(cuneibs::d_maxFluidBoundaryNeibs), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.maxVertexNeibs, HIP_SYMBOL(cuneibs::d_maxVertexNeibs), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.hasTooManyNeibs, HIP_SYMBOL(cuneibs::d_hasTooManyNeibs), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(timingInfo.hasMaxNeibs, HIP_SYMBOL(cuneibs::d_hasMaxNeibs), sizeof(int)*PT_TESTPOINT, 0));
}

/** @} */

/** \name Reordering and sort related function
 *  @{ */

/// Launch the compute hash kernel
/*!	Update the particle position and cell hash,
 * compute the particle index for sorting,
 * update the compact device map
 */
void
calcHash(	const BufferList& bufread, ///< input buffers (INFO, COMPACT_DEV_MAP)
			BufferList& bufwrite, ///< output buffers: HASH, POS (updated in place), PARTINDEX
			const uint	numParticles)			///< total number of particles
{
	uint numThreads = BLOCK_SIZE_CALCHASH;
	uint numBlocks = div_up(numParticles, numThreads);

	cuneibs::calcHashDevice<periodicbound><<< numBlocks, numThreads >>>
		(bufwrite.getData<BUFFER_POS>(),
		 bufwrite.getData<BUFFER_HASH>(),
		 bufwrite.getData<BUFFER_PARTINDEX>(),
		 bufread.getData<BUFFER_INFO>(),
		 bufread.getData<BUFFER_COMPACT_DEV_MAP>(),
		 numParticles);

	// Check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


/// Launch the fix hash kernel
/*!	Restricted version of \seealso calcHash, assuming the hash was already computed on host
 * and only needs a fixup to include the cell type specified in the COMPACT_DEV_MAP
 */
void
fixHash(	const BufferList& bufread, ///< input buffers (INFO, COMPACT_DEV_MAP)
			BufferList& bufwrite, ///< output buffers: HASH (updated in place), PARTINDEX
			const uint	numParticles)			///< total number of particles
{
	uint numThreads = BLOCK_SIZE_CALCHASH;
	uint numBlocks = div_up(numParticles, numThreads);

	cuneibs::fixHashDevice<<< numBlocks, numThreads >>>(
		bufwrite.getData<BUFFER_HASH>(),
		bufwrite.getData<BUFFER_PARTINDEX>(),
		bufread.getData<BUFFER_INFO>(),
		bufread.getData<BUFFER_COMPACT_DEV_MAP>(),
		numParticles);

	// Check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


/// Launch the reorder kernel
/*!	CPU part responsible of launching the reorder kernel
 * 	(cuneibs::reorderDataAndFindCellStartDevice) on the device.
 * 	\param[out] cellStart : index of cells first particle
 * 	\param[out] cellEnd : index of cells last particle
 * 	\param[out] segmentStart : TODO
 * 	\param[in] particleHash : sorted particle hashes
 * 	\param[in] particleIndex : sorted particle indices
 * 	\param[in] numParticles : total number of particles in input buffers
 * 	\param[out] newNumParticles : device pointer to number of active particles found
 */
void
reorderDataAndFindCellStart(
		uint*				segmentStart,		// TODO
		BufferList& sorted_buffers,			// list of sorted buffers (out)
		BufferList const& unsorted_buffers,	// list of buffers to sort (in)
		const uint			numParticles,		// total number of particles in input buffers (in)
		uint*				newNumParticles)	// device pointer to number of active particles found (out)
{

#if 0
#define MUST_HAVE(ar) \
	if (!ar) throw std::invalid_argument(#ar " is null")
#else
#define MUST_HAVE(ar) do { /* nothing */ } while (0)
#endif

#define BIND_CHECK(old_, new_, tex_) \
	if (old_) { \
		CUDA_SAFE_CALL(hipBindTexture(0, tex_, old_, numParticles*sizeof(*old_))); \
		MUST_HAVE(new_); \
	} \
	if (new_) MUST_HAVE(old_)

	const uint numThreads = BLOCK_SIZE_REORDERDATA;
	const uint numBlocks = div_up(numParticles, numThreads);

	const hashKey *particleHash = sorted_buffers.getConstData<BUFFER_HASH>();
	const uint *particleIndex = sorted_buffers.getConstData<BUFFER_PARTINDEX>();

	MUST_HAVE(particleHash);
	MUST_HAVE(particleIndex);

	// index of cells first and last particles (computed by the kernel)
	uint *cellStart = sorted_buffers.getData<BUFFER_CELLSTART>();
	uint *cellEnd = sorted_buffers.getData<BUFFER_CELLEND>();

	MUST_HAVE(cellStart);
	MUST_HAVE(cellEnd);

	// TODO find a smarter way to do this
	const float4 *oldPos = unsorted_buffers.getData<BUFFER_POS>();
	float4 *newPos = sorted_buffers.getData<BUFFER_POS>();
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, oldPos, numParticles*sizeof(float4)));
	MUST_HAVE(newPos);

	const float4 *oldVel = unsorted_buffers.getData<BUFFER_VEL>();
	float4 *newVel = sorted_buffers.getData<BUFFER_VEL>();
	CUDA_SAFE_CALL(hipBindTexture(0, velTex, oldVel, numParticles*sizeof(float4)));
	MUST_HAVE(newVel);

	const float4 *oldVol = unsorted_buffers.getData<BUFFER_VOLUME>();
	float4 *newVol = sorted_buffers.getData<BUFFER_VOLUME>();
	BIND_CHECK(oldVol, newVol, volTex);

	const float *oldEnergy = unsorted_buffers.getData<BUFFER_INTERNAL_ENERGY>();
	float *newEnergy = sorted_buffers.getData<BUFFER_INTERNAL_ENERGY>();
	BIND_CHECK(oldEnergy, newEnergy, energyTex);

	// sorted already
	const particleinfo *particleInfo = sorted_buffers.getConstData<BUFFER_INFO>();
	MUST_HAVE(particleInfo);

	const float4 *oldBoundElement = unsorted_buffers.getData<BUFFER_BOUNDELEMENTS>();
	float4 *newBoundElement = sorted_buffers.getData<BUFFER_BOUNDELEMENTS>();
	BIND_CHECK(oldBoundElement, newBoundElement, boundTex);

	const float4 *oldGradGamma = unsorted_buffers.getData<BUFFER_GRADGAMMA>();
	float4 *newGradGamma = sorted_buffers.getData<BUFFER_GRADGAMMA>();
	BIND_CHECK(oldGradGamma, newGradGamma, gamTex);

	const vertexinfo *oldVertices = unsorted_buffers.getData<BUFFER_VERTICES>();
	vertexinfo *newVertices = sorted_buffers.getData<BUFFER_VERTICES>();
	BIND_CHECK(oldVertices, newVertices, vertTex);

	const float *oldTKE = unsorted_buffers.getData<BUFFER_TKE>();
	float *newTKE = sorted_buffers.getData<BUFFER_TKE>();
	BIND_CHECK(oldTKE, newTKE, keps_kTex);

	const float *oldEps = unsorted_buffers.getData<BUFFER_EPSILON>();
	float *newEps = sorted_buffers.getData<BUFFER_EPSILON>();
	BIND_CHECK(oldEps, newEps, keps_eTex);

	const float *oldTurbVisc = unsorted_buffers.getData<BUFFER_TURBVISC>();
	float *newTurbVisc = sorted_buffers.getData<BUFFER_TURBVISC>();
	BIND_CHECK(oldTurbVisc, newTurbVisc, tviscTex);

	const float *oldEffPres = unsorted_buffers.getData<BUFFER_EFFPRES>();
	float *newEffPres = sorted_buffers.getData<BUFFER_EFFPRES>();
	if (oldEffPres)
		CUDA_SAFE_CALL(hipBindTexture(0, effpresTex, oldEffPres, numParticles*sizeof(float)));

	const float4 *oldEulerVel = unsorted_buffers.getData<BUFFER_EULERVEL>();
	float4 *newEulerVel = sorted_buffers.getData<BUFFER_EULERVEL>();
	BIND_CHECK(oldEulerVel, newEulerVel, eulerVelTex);

	const uint *oldNextIDs = unsorted_buffers.getData<BUFFER_NEXTID>();
	uint *newNextIDs = sorted_buffers.getData<BUFFER_NEXTID>();
	if (oldNextIDs && !newNextIDs)
		throw std::invalid_argument("newNextIDs is null");

	uint smemSize = sizeof(uint)*(numThreads+1);
	cuneibs::reorderDataAndFindCellStartDevice<<< numBlocks, numThreads, smemSize >>>(cellStart, cellEnd, segmentStart,
		newPos, newVel, newVol, newEnergy, newBoundElement, newGradGamma, newVertices, newTKE, newEps, newTurbVisc,
		newEffPres,
		newEulerVel,
		oldNextIDs, newNextIDs,
		particleInfo, particleHash, particleIndex, numParticles, newNumParticles);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	CUDA_SAFE_CALL(hipUnbindTexture(velTex));

	if (oldVol)
		CUDA_SAFE_CALL(hipUnbindTexture(volTex));
	if (oldEnergy)
		CUDA_SAFE_CALL(hipUnbindTexture(energyTex));

	if (oldBoundElement)
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	if (oldGradGamma)
		CUDA_SAFE_CALL(hipUnbindTexture(gamTex));
	if (oldVertices)
		CUDA_SAFE_CALL(hipUnbindTexture(vertTex));

	if (oldTKE)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_kTex));
	if (oldEps)
		CUDA_SAFE_CALL(hipUnbindTexture(keps_eTex));
	if (oldTurbVisc)
		CUDA_SAFE_CALL(hipUnbindTexture(tviscTex));

	if (oldEffPres)
		CUDA_SAFE_CALL(hipUnbindTexture(effpresTex));

	if (oldEulerVel)
		CUDA_SAFE_CALL(hipUnbindTexture(eulerVelTex));

#undef BIND_CHECK
#undef MUST_HAVE
}

/// Functor to sort particles by hash (cell), and
/// by fluid number within the cell
struct ptype_hash_compare :
	public thrust::binary_function<
		thrust::tuple<hashKey, particleinfo>,
		thrust::tuple<hashKey, particleinfo>,
		bool>
{
	typedef thrust::tuple<hashKey, particleinfo> value_type;

	__host__ __device__
	bool operator()(const value_type& a, const value_type& b)
	{
		const hashKey ha(cellHashFromParticleHash(thrust::get<0>(a), true)),
				hb(cellHashFromParticleHash(thrust::get<0>(b), true));
		const particleinfo pa(thrust::get<1>(a)),
					 pb(thrust::get<1>(b));

		if (ha == hb) {
			const ParticleType pta = PART_TYPE(pa),
				ptb = PART_TYPE(pb);
			if (pta == ptb)
				return id(pa) < id(pb);
			return (pta < ptb);
		}
		return (ha < hb);
	}
};

void
sort(	BufferList const& bufread,
		BufferList& bufwrite,
		uint	numParticles)
{
	thrust::device_ptr<particleinfo> particleInfo =
		thrust::device_pointer_cast(bufwrite.getData<BUFFER_INFO>());
	thrust::device_ptr<hashKey> particleHash =
		thrust::device_pointer_cast(bufwrite.getData<BUFFER_HASH>());
	thrust::device_ptr<uint> particleIndex =
		thrust::device_pointer_cast(bufwrite.getData<BUFFER_PARTINDEX>());

	ptype_hash_compare comp;

	if (numParticles > 0) {
		// Sort of the particle indices by cell, fluid number, id and
		// particle type (PT_FLUID < PT_BOUNDARY < PT_VERTEX)
		// There is no need for a stable sort due to the id sort
		thrust::sort_by_key(
			thrust::make_zip_iterator(thrust::make_tuple(particleHash, particleInfo)),
			thrust::make_zip_iterator(thrust::make_tuple(
				particleHash + numParticles,
				particleInfo + numParticles)),
			particleIndex, comp);
	}

	KERNEL_CHECK_ERROR;
}


/** @} */

/** \name Neighbors list building
 *  @{ */

/// Build neibs list
void
buildNeibsList(
const	BufferList&	bufread,
		BufferList&	bufwrite,
const	uint		numParticles,
const	uint		particleRangeEnd,
const	uint		gridCells,
const	float		sqinfluenceradius,
const	float		boundNlSqInflRad)
{
	const float4 *pos = bufread.getData<BUFFER_POS>();
	const particleinfo *info = bufread.getData<BUFFER_INFO>();
	const vertexinfo *vertices = bufread.getData<BUFFER_VERTICES>();
	const float4 *boundelem = bufread.getData<BUFFER_BOUNDELEMENTS>();
	const hashKey *particleHash = bufread.getData<BUFFER_HASH>();
	const uint *cellStart = bufread.getData<BUFFER_CELLSTART>();
	const uint *cellEnd = bufread.getData<BUFFER_CELLEND>();

	neibdata	*neibsList = bufwrite.getData<BUFFER_NEIBSLIST>();
	float2		**vertPos  = bufwrite.getRawPtr<BUFFER_VERTPOS>();

	// vertices, boundeleme and vertPos must be either all NULL or all not-NULL.
	// throw otherwise
	if (vertices || boundelem || vertPos) {
		if (!vertices || !boundelem || ! vertPos) {
			fprintf(stderr, "%p vs %p vs %p\n", vertices, boundelem, vertPos);
			throw std::invalid_argument("inconsistent params to buildNeibsList");
		}
	}

	if (boundarytype == SA_BOUNDARY && !vertices) {
		fprintf(stderr, "%s boundary type selected, but no vertices!\n",
			BoundaryName[boundarytype]);
		throw std::invalid_argument("missing data");
	}

	const uint numThreads = BLOCK_SIZE_BUILDNEIBS;
	const uint numBlocks = div_up(particleRangeEnd, numThreads);

	// bind textures to read all particles, not only internal ones
	#if !PREFER_L1
	CUDA_SAFE_CALL(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
	#endif
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));
	CUDA_SAFE_CALL(hipBindTexture(0, cellStartTex, cellStart, gridCells*sizeof(uint)));
	CUDA_SAFE_CALL(hipBindTexture(0, cellEndTex, cellEnd, gridCells*sizeof(uint)));

	if (boundarytype == SA_BOUNDARY) {
		CUDA_SAFE_CALL(hipBindTexture(0, vertTex, vertices, numParticles*sizeof(vertexinfo)));
		CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelem, numParticles*sizeof(float4)));
	}

	buildneibs_params<boundarytype> params(neibsList, pos, particleHash, particleRangeEnd, sqinfluenceradius,
			vertPos, boundNlSqInflRad);

	cuneibs::buildNeibsListDevice<sph_formulation, /*ViscSpec, */boundarytype, periodicbound, neibcount><<<numBlocks, numThreads>>>(params);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	if (boundarytype == SA_BOUNDARY) {
		CUDA_SAFE_CALL(hipUnbindTexture(vertTex));
		CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	}

	#if !PREFER_L1
	CUDA_SAFE_CALL(hipUnbindTexture(posTex));
	#endif
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));
	CUDA_SAFE_CALL(hipUnbindTexture(cellStartTex));
	CUDA_SAFE_CALL(hipUnbindTexture(cellEndTex));
}

/** @} */

};

