#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

/*
* Device code.
*/

#ifndef _DENSITY_SUM_KERNEL_
#define _DENSITY_SUM_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "multi_gpu_defines.h"

namespace cudensity_sum {

using namespace cusph;
using namespace cuphys;
using namespace cuneibs;
using namespace cueuler;

struct density_sum_particle_output
{
float4	gGamNp1;
float	rho;

__device__ __forceinline__
density_sum_particle_output() :
	gGamNp1(make_float4(0.0f)),
	rho(0.0f)
{}
};

struct common_density_sum_particle_data
{
const	uint	index;
const	particleinfo	info;
const	ParticleType	ptype;
const	float4	force;
const	int3	gridPos;
const	float4	posN;
const	float4	posNp1;
const	float4	vel;
const	float4	gGamN;

__device__ __forceinline__
common_density_sum_particle_data(const uint _index, common_density_sum_params const& params) :
	index(_index),
	info(params.info[index]),
	ptype(static_cast<ParticleType>(PART_TYPE(info))),
	force(params.forces[index]),
	gridPos(calcGridPosFromParticleHash(params.particleHash[index])),
	posN(params.oldPos[index]),
	posNp1(params.newPos[index]),
	vel(params.oldVel[index]),
	gGamN(params.oldgGam[index])
{}
};

struct open_boundary_particle_data
{
const	float4	eulerVel;

__device__ __forceinline__
open_boundary_particle_data(const uint index, EulerVel_params<false> const& params) :
	eulerVel(params.oldEulerVel[index])
{}
};

/// The actual density_sum_particle_data struct, which concatenates all of the above, as appropriate.
template<KernelType _kerneltype,
ParticleType _ntype,
flag_t _simflags>
struct density_sum_particle_data :
common_density_sum_particle_data,
COND_STRUCT((_simflags & ENABLE_INLET_OUTLET),
			open_boundary_particle_data)
{
static const KernelType kerneltype = _kerneltype;
static const ParticleType ntype = _ntype;
static const flag_t simflags = _simflags;

// shorthand for the type of the density_sum params
typedef density_sum_params<kerneltype, ntype, simflags> params_t;

// determine specialization automatically based on info and params
__device__ __forceinline__
density_sum_particle_data(const uint _index, params_t const& params) :
	common_density_sum_particle_data(_index, params),
	COND_STRUCT((_simflags & ENABLE_INLET_OUTLET),
				open_boundary_particle_data)(_index, params)
{}
};

template<SPHFormulation sph_formulation, class Params, class ParticleData, KernelType kerneltype=ParticleData::kerneltype>
__device__ __forceinline__
enable_if_t<	(Params::simflags & ENABLE_INLET_OUTLET) &&
	sph_formulation != SPH_HA>
densitySumOpenBoundaryContribution(
Params			const&	params,
ParticleData	const&	pdata,
const	float	dt,
const	uint	neib_index,
const	particleinfo neib_info,
const	float4&	relPosN,
float&	sumVmwDelta)
{
if (IO_BOUNDARY(neib_info)) {
	// compute - sum_{V^{io}} m^n w(r + delta r)
	const float4 deltaR = dt*(params.oldEulerVel[neib_index] - params.oldVel[neib_index]);
	const float newDist = length3(relPosN + deltaR);
	if (newDist < params.influenceradius)
		sumVmwDelta -= relPosN.w*W<kerneltype>(newDist, params.slength);
}
}

// Partial specialization for SPH_HA:
// the only difference is that the current fluid particle mass
// is used instead of the neib mass. This comes from the fact that we use:
// rho_i = mi ∑ wij   instead of    rho_i = ∑ mj.wij
template<SPHFormulation sph_formulation, class Params, class ParticleData, KernelType kerneltype=ParticleData::kerneltype>
__device__ __forceinline__
enable_if_t<	(Params::simflags & ENABLE_INLET_OUTLET) &&
	sph_formulation == SPH_HA>
densitySumOpenBoundaryContribution
(
Params			const&	params,
ParticleData	const&	pdata,
const	float	dt,
const	uint	neib_index,
const	particleinfo neib_info,
const	float4&	relPosN,
float&	sumVmwDelta,
const	float	thetaRatio_times_pmass)
{
// TODO: IO were implemented but not tested with Hu & Adams formulation
if (IO_BOUNDARY(neib_info)) {
	// compute - sum_{V^{io}} m^n w(r + delta r)
	const float4 deltaR = dt*(params.oldEulerVel[neib_index] - params.oldVel[neib_index]);
	const float newDist = length3(relPosN + deltaR);
	if (newDist < params.influenceradius)
		sumVmwDelta -= thetaRatio_times_pmass*W<kerneltype>(newDist, params.slength);
}
}

template<SPHFormulation sph_formulation, class Params, class ParticleData, KernelType kerneltype=ParticleData::kerneltype>
__device__ __forceinline__
enable_if_t<	!(Params::simflags & ENABLE_INLET_OUTLET) &&
	sph_formulation != SPH_HA>
densitySumOpenBoundaryContribution(
Params			const&	params,
ParticleData	const&	pdata,
const	float	dt,
const	uint	neib_index,
const	particleinfo neib_info,
const	float4&	relPosN,
float&	sumVmwDelta)
{ /* do nothing */ }

template<SPHFormulation sph_formulation, class Params, class ParticleData, KernelType kerneltype=ParticleData::kerneltype>
__device__ __forceinline__
enable_if_t<	!(Params::simflags & ENABLE_INLET_OUTLET) &&
	sph_formulation == SPH_HA>
densitySumOpenBoundaryContribution(
Params			const&	params,
ParticleData	const&	pdata,
const	float	dt,
const	uint	neib_index,
const	particleinfo neib_info,
const	float4&	relPosN,
float&	sumVmwDelta,
float	thetaRatio_times_pmass)
{ /* do nothing */ }


template<SPHFormulation sph_formulation, class Params, class ParticleData, KernelType kerneltype=ParticleData::kerneltype>
__device__ __forceinline__
static
enable_if_t<sph_formulation != SPH_HA>
computeDensitySumVolumicTerms(
Params			const&	params,
ParticleData	const&	pdata,
const	float			dt,
		float			&sumPmwN,
		float			&sumPmwNp1,
		float			&sumVmwDelta)
{
// Compute grid position of current particle
const int3 gridPos = calcGridPosFromParticleHash( params.particleHash[ pdata.index] );

// (r_b^{N+1} - r_b^N)
const float4 posDelta = pdata.posNp1 - pdata.posN;

// Loop over fluid and vertex neighbors
for_each_neib2(PT_FLUID, PT_VERTEX, pdata.index, pdata.posN, gridPos, params.cellStart, params.neibsList) {
	const uint neib_index = neib_iter.neib_index();
	const particleinfo neib_info = params.info[neib_index];

	const float4 posN_neib = params.oldPos[neib_index];

	if (INACTIVE(posN_neib)) continue;

	/* TODO FIXME splitneibs merge: the MOVING object support here was dropped in the splitneibs branch */

	const float4 posNp1_neib = params.newPos[neib_index];

	// vector r_{ab} at time N
	const float4 relPosN = neib_iter.relPos(posN_neib);
	// vector r_{ab} at time N+1 = r_{ab}^N + (r_a^{N+1} - r_a^{N}) - (r_b^{N+1} - r_b^N)
	const float4 relPosNp1 = neib_iter.relPos(posNp1_neib) + posDelta;

	// -sum_{P\V_{io}} m^n w^n
	if (!IO_BOUNDARY(neib_info)) {
		const float rN = length3(relPosN);
		sumPmwN -= relPosN.w*W<kerneltype>(rN, params.slength);
	}

	// sum_{P} m^n w^{n+1}
	const float rNp1 = length3(relPosNp1);
	if (rNp1 < params.influenceradius)
		sumPmwNp1 += relPosN.w*W<kerneltype>(rNp1, params.slength);

		if (!params.repacking)
			densitySumOpenBoundaryContribution<sph_formulation>(params, pdata, dt,
				neib_index, neib_info, relPosN, sumVmwDelta);
	}
}

// Partial specialization for SPH_HA
// the only difference is that the current fluid particle mass
// is used instead of the neib mass. This comes from the fact that we use:
// rho_i = mi/theta_i ∑ theta_j.wij   instead of    rho_i = ∑ mj.wij
template<SPHFormulation sph_formulation, class Params, class ParticleData, KernelType kerneltype=ParticleData::kerneltype>
__device__ __forceinline__
static
enable_if_t<sph_formulation == SPH_HA>
computeDensitySumVolumicTerms(
	Params			const&	params,
	ParticleData	const&	pdata,
	const	float			dt,
			float			&sumPmwN,
			float			&sumPmwNp1,
			float			&sumVmwDelta)
{
	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( params.particleHash[ pdata.index] );

	// (r_b^{N+1} - r_b^N)
	const float4 posDelta = pdata.posNp1 - pdata.posN;

	// Get particleinfo of the current particle
	const particleinfo p_info = params.info[ pdata.index ];
	// Get the fluid number of the current particle
	const uint p_fluid_num = fluid_num(p_info);

	// Loop over fluid and vertex neighbors
	for_each_neib2(PT_FLUID, PT_VERTEX, pdata.index, pdata.posN, gridPos, params.cellStart, params.neibsList) {
		const uint neib_index = neib_iter.neib_index();
		const particleinfo neib_info = params.info[neib_index];

		// Get the fluid number of neib particle
		const uint neib_fluid_num = fluid_num(neib_info);

		const float4 posN_neib = params.oldPos[neib_index];
		//const float4 velN_neib = params.oldVel[neib_index];

		if (INACTIVE(posN_neib)) continue;

		/* TODO FIXME splitneibs merge: the MOVING object support here was dropped in the splitneibs branch */

		const float4 posNp1_neib = params.newPos[neib_index];

		// vector r_{ab} at time N
		const float4 relPosN = neib_iter.relPos(posN_neib);
		// vector r_{ab} at time N+1 = r_{ab}^N + (r_a^{N+1} - r_a^{N}) - (r_b^{N+1} - r_b^N)
		const float4 relPosNp1 = neib_iter.relPos(posNp1_neib) + posDelta;

		const float p_volume0 = pdata.posN.w/d_rho0[p_fluid_num];
		const float n_volume0 = relPosN.w/d_rho0[neib_fluid_num];
		const float p_theta = p_volume0/(params.deltap*params.deltap*params.deltap);
		const float n_theta = n_volume0/(params.deltap*params.deltap*params.deltap);
		const float thetaRatio_times_pmass = pdata.posN.w/p_theta*n_theta;

		// -sum_{P\V_{io}} m^n w^n
		if (!IO_BOUNDARY(neib_info)) {
			const float rN = length3(relPosN);
			sumPmwN -= thetaRatio_times_pmass*W<kerneltype>(rN, params.slength);
		}

		// sum_{P} m^n w^{n+1}
		const float rNp1 = length3(relPosNp1);
		if (rNp1 < params.influenceradius) {
			sumPmwNp1 += thetaRatio_times_pmass*W<kerneltype>(rNp1, params.slength);
		}
		densitySumOpenBoundaryContribution<SPH_HA>(params, pdata, dt,
			neib_index, neib_info, relPosN, sumVmwDelta, thetaRatio_times_pmass);
	}
}


struct common_gamma_sum_terms {
	// collects sum_{S} (gradGam^{n+1} + gradGam^n)/2 . (r^{n+1} - r^{n})
	float gGamDotR;
	// gradGam
	float3 gGam;

	__device__ __forceinline__
	common_gamma_sum_terms() :
		gGamDotR(0.0f),
		gGam(make_float3(0.0f))
	{}
};

/// Gamma summation terms in case of I/O
struct io_gamma_sum_terms {
	// sum_{S^{io}} (gradGam(r + delta r)).delta r
	float sumSgamDelta;
	// sum_{S^{io}} (gradGam(r)).delta r
	float sumSgamN;

	__device__ __forceinline__
	io_gamma_sum_terms() :
		sumSgamDelta(0.0f),
		sumSgamN(0.0f)
	{}
};

template<KernelType _kerneltype,
	flag_t simflags,
	bool _has_io = !!(simflags & ENABLE_INLET_OUTLET)>
struct gamma_sum_terms :
	common_gamma_sum_terms,
	COND_STRUCT(_has_io, io_gamma_sum_terms)
{
	static constexpr KernelType kerneltype = _kerneltype;
	static constexpr bool has_io = _has_io;
};

/* contribution to grad gamma integration from I/O,
 * only if I/O is active
 */
template<typename Params, typename GammaTermT>
__device__ __forceinline__
enable_if_t<!GammaTermT::has_io>
io_gamma_contrib(GammaTermT &sumGam, int neib_index, particleinfo const& neib_info,
	Params const& params,
	const float3 qN,
	const float3 ns,
	const float3 * vertexRelPos,
	float dt,
	const float3	gGamN)
{ /* default case (no I/O), nothing to do */ };

template<typename Params, typename GammaTermT>
__device__ __forceinline__
enable_if_t<GammaTermT::has_io>
io_gamma_contrib(GammaTermT &sumGam, int neib_index, particleinfo const& neib_info,
	Params const& params,
	const float3 qN,
	const float3 ns,
	const float3 * vertexRelPos,
	float dt,
	const float3	gGamN)
{
		if (IO_BOUNDARY(neib_info)) {
			// sum_{S^{io}} (gradGam(r + delta r)).delta r
			const float3 deltaR = dt*as_float3(params.oldEulerVel[neib_index] - params.oldVel[neib_index]);
			const float3 qDelta = qN + deltaR/params.slength;
			const float3 gGamDelta = gradGamma<GammaTermT::kerneltype>(params.slength, qDelta, vertexRelPos, ns)*ns;
			sumGam.sumSgamDelta += dot(deltaR, gGamDelta);
			sumGam.sumSgamN += dot(deltaR, gGamN);
		}
};

// Compute the imposedGamma for densitySumBoundaryDevice, depending on IO conditions
template<typename GammaTermT>
__device__ __forceinline__
enable_if_t<!GammaTermT::has_io, float>
compute_imposed_gamma(float oldGam, GammaTermT const& sumGam)
{
	return oldGam;
}
template<typename GammaTermT>
__device__ __forceinline__
enable_if_t<GammaTermT::has_io, float>
compute_imposed_gamma(float oldGam, GammaTermT const& sumGam)
{
	float imposed = oldGam + (sumGam.sumSgamDelta + sumGam.sumSgamN)/2.0f;
	// clipping of the imposed gamma
	if (imposed > 1.0f)
		imposed = 1.0f;
	else if (imposed < 0.1f)
		imposed = 0.1f;

	return imposed;
}

template<class Params, class ParticleData,
	KernelType kerneltype=ParticleData::kerneltype,
	flag_t simflags=ParticleData::simflags>
__device__ __forceinline__
static void
computeDensitySumBoundaryTerms(
	Params			const&	params,
	ParticleData	const&	pdata,
	const	float			dt,
	gamma_sum_terms<kerneltype, simflags> &sumGam)
{
	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( params.particleHash[pdata.index] );

	// (r_b^{N+1} - r_b^N)
	const float4 posDelta = pdata.posNp1 - pdata.posN;

	// Loop over BOUNDARY neighbors
	for_each_neib(PT_BOUNDARY, pdata.index, pdata.posN, gridPos, params.cellStart, params.neibsList) {
		const uint neib_index = neib_iter.neib_index();
		const particleinfo neib_info = params.info[neib_index];

		const float4 posN_neib = params.oldPos[neib_index];

		if (INACTIVE(posN_neib)) continue;

		const float4 posNp1_neib = params.newPos[neib_index];

		// vector r_{ab} at time N
		const float4 qN = neib_iter.relPos(posN_neib)/params.slength;
		// vector r_{ab} at time N+1 = r_{ab}^N + (r_a^{N+1} - r_a^{N}) - (r_b^{N+1} - r_b^N)
		const float4 qNp1 = (neib_iter.relPos(posNp1_neib) + posDelta)/params.slength;

		float3 vertexRelPos[3];

		// normal of segment
		const float3 nsN = make_float3(params.oldBoundElement[neib_index]);
		calcVertexRelPos(vertexRelPos, nsN,
			params.vertPos0[neib_index], params.vertPos1[neib_index], params.vertPos2[neib_index],
			params.slength);
		const float3 gGamN   = gradGamma<kerneltype>(params.slength, as_float3(qN),   vertexRelPos, nsN)*nsN;

		const float3 nsNp1 = make_float3(params.newBoundElement[neib_index]);
		/* We only need to recompute calcVertexRelPos wrt to the new normal if there are moving bodies,
		 * since otherwise the new normal is the same as the old normal */
		if (simflags & ENABLE_MOVING_BODIES)
			calcVertexRelPos(vertexRelPos, nsNp1,
				params.vertPos0[neib_index], params.vertPos1[neib_index], params.vertPos2[neib_index],
				params.slength);
		/* But we still need to reocmpute grad gamma, because q changed anyway */
		const float3 gGamNp1 = gradGamma<kerneltype>(params.slength, as_float3(qNp1), vertexRelPos, nsNp1)*nsNp1;

		// sum_S 1/2*(gradGam^n + gradGam^{n+1})*relVel
		sumGam.gGamDotR += 0.5f*dot(gGamN + gGamNp1, as_float3(qNp1 - qN));
		sumGam.gGam += gGamNp1;

		/* TODO check if we need the old or the new normal here, in case of
		 * moving open boundaries (for fixed open boundaries, it makes no difference)
		 */
		if (!params.repacking)
			io_gamma_contrib(sumGam, neib_index, neib_info, params,
				make_float3(qN), nsN, vertexRelPos, dt, gGamN);
	}
	sumGam.gGamDotR *= params.slength;
}

/// Computes the density based on an integral formulation of the continuity equation
/*! Updates the density of fluid particles
 *
 *	\param[in] oldPos : previous particle's position
 *	\param[in] hashKey : particle's hash
 *	\param[in] oldVel : previous particle's velocity
 *	\param[in] oldEulerVel : previous eulerian velocities for ??? <- TODO
 *	\param[in] oldGam : previous values of gradient of gamma
 *	\param[in] okdTKE : previous values of k, for k-e model
 *	\param[in] oldEps : previous values of e, for k-e model
 *	\param[in] particleInfo : particle's information
 *	\param[in] forces : derivative of particle's velocity and density
 *	\param[in] keps_dkde : derivative of ??? <- TODO
 *	\param[in] xsph : SPH mean of velocities used for xsph correction
 *	\param[out] newPos : updated particle's position
 *	\param[out] newVel : updated particle's  velocity
 *	\param[out] newEulerVel : updated eulerian velocities for ??? <- TODO
 *	\param[out] newgGam : updated values of gradient of gamma
 *	\param[out] newTKE : updated values of k, for k-e model
 *	\param[out] newEps : updated values of e, for k-e model
 *	\param[in,out] newBoundElement : ??? <- TODO
 *	\param[in] numParticles : total number of particles
 *	\param[in] dt  : time step (dt or dt/2, depending on the integration step)
 *	\param[in] t : simualation time
 *
 *	\tparam step : integration step (1, 2)
 *	\tparam boundarytype : type of boundary
 *	\tparam kerneltype : type of kernel
 *	\tparam simflags : simulation flags
 */
//TODO templatize vars like other kernels
template<SPHFormulation sph_formulation,
	KernelType kerneltype,
	flag_t simflags>
__global__ void
densitySumVolumicDevice(
	// parameters are the same for fluid and vertex
	density_sum_params<kerneltype, PT_FLUID, simflags> params)
{
	const int index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	// only perform density integration for fluid particles
	if (index >= params.numParticles || !FLUID(params.info[index]))
		return;

	// this is either dt or dt/2, depending on the integrator step
	const float dt = params.dt;

	density_sum_particle_data<kerneltype, PT_FLUID, simflags> pdata(index, params);

	density_sum_particle_output pout;

	// continuity equation based on particle positions
	// - sum_{P\V^{io}} m^n w^n
	float sumPmwN = 0.0f;
	// sum_{P} m^n w^{n+1}
	float sumPmwNp1 = 0.0f;
	// - sum_{V^{io}} m^n w(r + delta r)
	float sumVmwDelta = 0.0f;
	// compute new terms based on r^{n+1} and \delta r
	computeDensitySumVolumicTerms<sph_formulation>(
		params, pdata, dt,
		sumPmwN, sumPmwNp1, sumVmwDelta);

	params.forces[index].w = sumPmwNp1 + sumPmwN + sumVmwDelta;
}

struct integrate_gamma_particle_data
{
	const	uint	index;
	const	float4	posN;
	const	float4	posNp1;

	template<typename Params>
	__device__ __forceinline__
	integrate_gamma_particle_data(const uint _index, Params const& params) :
		index(_index),
		posN(params.oldPos[index]),
		posNp1(params.newPos[index])
	{}
};

/// Computes the density based on an integral formulation of the continuity equation
/*! Updates the density of fluid particles
 *
 *	\param[in] oldPos : previous particle's position
 *	\param[in] hashKey : particle's hash
 *	\param[in] oldVel : previous particle's velocity
 *	\param[in] oldEulerVel : previous eulerian velocities for ??? <- TODO
 *	\param[in] oldGam : previous values of gradient of gamma
 *	\param[in] okdTKE : previous values of k, for k-e model
 *	\param[in] oldEps : previous values of e, for k-e model
 *	\param[in] particleInfo : particle's information
 *	\param[in] forces : derivative of particle's velocity and density
 *	\param[in] dgamdt : time derivative of gamma
 *	\param[in] keps_dkde : derivative of ??? <- TODO
 *	\param[in] xsph : SPH mean of velocities used for xsph correction
 *	\param[out] newPos : updated particle's position
 *	\param[out] newVel : updated particle's  velocity
 *	\param[out] newEulerVel : updated eulerian velocities for ??? <- TODO
 *	\param[out] newgGam : updated values of gradient of gamma
 *	\param[out] newTKE : updated values of k, for k-e model
 *	\param[out] newEps : updated values of e, for k-e model
 *	\param[in,out] newBoundElement : ??? <- TODO
 *	\param[in] numParticles : total number of particles
 *	\param[in] full_dt  : time step (dt)
 *	\param[in] half_dt : half of time step (dt/2)
 *	\param[in] t : simualation time
 *
 *	\tparam step : integration step (1, 2)
 *	\tparam boundarytype : type of boundary
 *	\tparam kerneltype : type of kernel
 *	\tparam simflags : simulation flags
 */
//TODO templatize vars like other kernels
template<KernelType kerneltype,
	flag_t simflags>
__global__ void
densitySumBoundaryDevice(
	density_sum_params<kerneltype, PT_BOUNDARY, simflags> params)
{
	const int index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	// only perform density integration for fluid particles
	if (index >= params.numParticles || !FLUID(params.info[index]))
		return;

	// this is either dt or dt/2, depending on the integrator step
	const float dt = params.dt;

	density_sum_particle_data<kerneltype, PT_BOUNDARY, simflags> pdata(index, params);

	density_sum_particle_output pout;

	// continuity equation based on particle positions

	gamma_sum_terms<kerneltype, simflags> sumGam;

	computeDensitySumBoundaryTerms(params, pdata, dt, sumGam);

	pout.gGamNp1.x = sumGam.gGam.x;
	pout.gGamNp1.y = sumGam.gGam.y;
	pout.gGamNp1.z = sumGam.gGam.z;

	// gamma terms
	// AM-TODO what about this term to remove 1/2 dgamdt?
	//const float4 gGamN = pdata.newgGam;// - (step-1.0)*make_float4(0.0f, 0.0f, 0.0f, gGamDotR/2.0f);
	pout.gGamNp1.w = pdata.gGamN.w + sumGam.gGamDotR;

	// now compute a new gamma based on the eulerian velocity of the boundary
	float imposedGam = compute_imposed_gamma(pdata.gGamN.w, sumGam);

	// generate new density based on previously computed values
	pout.rho = (imposedGam*physical_density(pdata.vel.w,fluid_num(pdata.info)) + params.forces[index].w)/pout.gGamNp1.w;

	// clipping of new gamma
	// this needs to happen after the density update because otherwise density jumps can occur
	if (pout.gGamNp1.w > 1.0f || length3(pout.gGamNp1)*params.slength < 1e-10f)
		pout.gGamNp1.w = 1.0f;
	else if (pout.gGamNp1.w < 0.1f)
		pout.gGamNp1.w = 0.1f;

	// output of updated variables:
	// density
	params.newVel[index].w = numerical_density(pout.rho,fluid_num(pdata.info));
	// gamma
	params.newgGam[index] = pout.gGamNp1;
}

/// Integrate gamma
/** We need two specializations of this kernel, one for gamma quadrature case,
 * and one for the dynamic gamma case. Since we can't use enable_if to select
 * the case, we refactor the specialization into an integrateGammaDeviceFunc
 * called by the kernel directly.
 *
 * The dynamic gamma case uses the same approach as the density sum,
 * computing gamma from the difference of the particle distribution at step n
 * and at step n+1 (hence why the kernel is here in the density sum namespace).
 *
 * The quadrature case computes gamma from the quadrature formula directly.
 */
template<typename Params>
__device__ __forceinline__
enable_if_t<USING_DYNAMIC_GAMMA(Params::simflags)>
integrateGammaDeviceFunc(Params params, uint index)
{
	// this is dt or dt/2, depending on the integrator step
	const float dt = params.dt;

	integrate_gamma_particle_data pdata(index, params);

	gamma_sum_terms<Params::kerneltype, Params::simflags> sumGam;

	computeDensitySumBoundaryTerms(params, pdata, dt, sumGam);

	params.newgGam[index] = make_float4(sumGam.gGam, params.oldgGam[index].w + sumGam.gGamDotR);
}

struct quadrature_gamma_particle_data
{
	const float4	oldGGam;
	const float4	pos;
	const int3	gridPos;

	template<typename FP>
	__device__ __forceinline__
	quadrature_gamma_particle_data(FP const& params, uint index) :
		oldGGam(params.oldgGam[index]),
		pos(params.newPos[index]),
		gridPos(calcGridPosFromParticleHash(params.particleHash[index]))
	{}
};

struct quadrature_gamma_particle_output
{
	float4 gGam;

	__device__ __forceinline__
	quadrature_gamma_particle_output() :
		gGam(make_float4(0, 0, 0, 1))
	{}
};

struct quadrature_gamma_neib_data
{
	const uint index;
	const float4 relPos;
	const float4 belem;

	template<typename FP, typename Iterator>
	__device__ __forceinline__
	quadrature_gamma_neib_data(FP const& params, Iterator const& iter) :
		index(iter.neib_index()),
		relPos(iter.relPos(params.newPos[index])),
		belem(params.newBoundElement[index])
	{}

};

/// Contribution to gamma and gamma gradient from a single neighbor, in the case of gamma quadrature
template<typename FP, typename P, typename N, typename OP>
__device__ __forceinline__
void
gamma_quadrature_contrib(FP const& params, P const& pdata, N const& ndata, OP &pout)
{
	const float3 q = as_float3(ndata.relPos)/params.slength;
	float3 q_vb[3];
	calcVertexRelPos(q_vb, ndata.belem,
		params.vertPos0[ndata.index], params.vertPos1[ndata.index], params.vertPos2[ndata.index],
		params.slength);

	float ggamAS = gradGamma<FP::kerneltype>(params.slength, q, q_vb, as_float3(ndata.belem));
	pout.gGam.x += ggamAS*ndata.belem.x;
	pout.gGam.y += ggamAS*ndata.belem.y;
	pout.gGam.z += ggamAS*ndata.belem.z;

	const float gamma_as = Gamma<FP::kerneltype, FP::cptype>(params.slength, q, q_vb, as_float3(ndata.belem),
		as_float3(pdata.oldGGam), params.epsilon);
	pout.gGam.w -= gamma_as;
}

/// Integrate gamma using gamma quadrature
template<typename Params>
__device__ __forceinline__
enable_if_t<!USING_DYNAMIC_GAMMA(Params::simflags)>
integrateGammaDeviceFunc(Params params, const uint index)
{
	const quadrature_gamma_particle_data pdata(params, index);
	quadrature_gamma_particle_output pout;

	for_each_neib(PT_BOUNDARY, index, pdata.pos, pdata.gridPos,
		params.cellStart, params.neibsList)
	{
		const quadrature_gamma_neib_data ndata(params, neib_iter);

		gamma_quadrature_contrib(params, pdata, ndata, pout);
	}

	params.newgGam[index] = pout.gGam;
}

template<typename Params>
__global__ void
integrateGammaDevice(Params params)
{
	const int index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= params.particleRangeEnd)
		return;

	const particleinfo pinfo = params.info[index];

	/* We only need to integrate gamma on fluid and vertex particles */
	/* And actually vertex particles should only be considered in the case
	 * of moving bodies or open boundaries */
	if (PART_TYPE(pinfo) != Params::cptype)
		return;

	integrateGammaDeviceFunc(params, index);
}

} // end of namespace cudensity_sum
#endif
