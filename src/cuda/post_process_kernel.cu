#include "hip/hip_runtime.h"
/*  Copyright (c) 2013-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

/*
 * Device code.
 */

#ifndef _POST_PROCESS_KERNEL_
#define _POST_PROCESS_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "vector_math.h"
#include "multi_gpu_defines.h"
#include "GlobalData.h"

#if __COMPUTE__ < 20
#define printf(...) /* eliminate printf from 1.x */
#endif

namespace cupostprocess {

using namespace cugeom;
using namespace cusph;
using namespace cuphys;
using namespace cuneibs;
using namespace hipcubounds;

/************************************************************************************************************/
/*					   Auxiliary kernels used for post processing										    */
/************************************************************************************************************/

//! Computes the vorticity field
template<KernelType kerneltype>
__global__ void
calcVortDevice(	const	float4*		posArray,
						float3*		vorticity,
				const	hashKey*		particleHash,
				const	uint*		cellStart,
				const	neibdata*	neibsList,
				const	uint		numParticles,
				const	float		slength,
				const	float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// computing vorticity only for active fluid particles
	const particleinfo info = tex1Dfetch(infoTex, index);

	#if PREFER_L1
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	if (NOT_FLUID(info) || INACTIVE(pos)) {
		vorticity[index] = make_float3(NAN);
		return;
	}

	const float4 vel = tex1Dfetch(velTex, index);

	float3 vort = make_float3(0.0f);

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// First loop over all FLUID neighbors
	for_each_neib(PT_FLUID, index, pos, gridPos, cellStart, neibsList) {
		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length(as_float3(relPos));

		// Compute relative velocity
		// Now relVel is a float4 and neib density is stored in relVel.w
		const float4 relVel = as_float3(vel) - tex1Dfetch(velTex, neib_index);
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Compute vorticity
		if (r < influenceradius) {
			const float f = F<kerneltype>(r, slength)*relPos.w/physical_density(relVel.w,fluid_num(neib_info));	// ∂Wij/∂r*Vj
			// vxij = vxi - vxj and same for vyij and vzij
			vort.x += f*(relVel.y*relPos.z - relVel.z*relPos.y);		// vort.x = ∑(vyij(zi - zj) - vzij*(yi - yj))*∂Wij/∂r*Vj
			vort.y += f*(relVel.z*relPos.x - relVel.x*relPos.z);		// vort.y = ∑(vzij(xi - xj) - vxij*(zi - zj))*∂Wij/∂r*Vj
			vort.z += f*(relVel.x*relPos.y - relVel.y*relPos.x);		// vort.x = ∑(vxij(yi - yj) - vyij*(xi - xj))*∂Wij/∂r*Vj
		}
	} // end of loop trough neighbors

	vorticity[index] = vort;
}


//! Compute the values of velocity, density, k and epsilon at test points
template<KernelType kerneltype,
	BoundaryType boundarytype>
__global__ void
calcTestpointsVelocityDevice(	const float4*	oldPos,
								float4*			newVel,
								float*			newTke,
								float*			newEpsilon,
								const hashKey*	particleHash,
								const uint*		cellStart,
								const neibdata*	neibsList,
								const uint		numParticles,
								const float		slength,
								const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	const particleinfo info = tex1Dfetch(infoTex, index);
	if(!TESTPOINT(info))
		return;

	#if PREFER_L1
	const float4 pos = oldPos[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// this is the velocity (x,y,z) and pressure (w)
	float4 velavg = make_float4(0.0f);
	// this is for k/epsilon
	float tkeavg = 0.0f;
	float epsavg = 0.0f;
	// this is the shepard filter sum(w_b w_{ab})
	float alpha = 0.0f;

	// Compute grid position of current particle
	int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// First loop over FLUID and VERTEX neighbors (VERTEX only in SA case)
	for_each_neib2(PT_FLUID, (boundarytype == SA_BOUNDARY ? PT_VERTEX : PT_NONE),
			index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			oldPos[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		const float r = length(as_float3(relPos));

		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius) {
			const float4 neib_vel = tex1Dfetch(velTex, neib_index);
			const float w = W<kerneltype>(r, slength)*relPos.w/physical_density(neib_vel.w,fluid_num(neib_info));	// Wij*mj
			//Velocity
			velavg.x += w*neib_vel.x;
			velavg.y += w*neib_vel.y;
			velavg.z += w*neib_vel.z;
			//Pressure
			velavg.w += w*P(neib_vel.w, fluid_num(neib_info));
			// Turbulent kinetic energy
			if(newTke){
				const float neib_tke = tex1Dfetch(keps_kTex, neib_index);
				tkeavg += w*neib_tke;
			}
			if(newEpsilon){
				const float neib_eps = tex1Dfetch(keps_eTex, neib_index);
				epsavg += w*neib_eps;
			}
			//Shepard filter
			alpha += w;
		}
	}

	// Renormalization by the Shepard filter
	if(alpha>1e-5f) {
		velavg /= alpha;
		if(newTke)
			tkeavg /= alpha;
		if(newEpsilon)
			epsavg /= alpha;
	}
	else {
		velavg = make_float4(0.0f);
		if(newTke)
			tkeavg = 0.0f;
		if(newEpsilon)
			epsavg = 0.0f;
	}

	newVel[index] = velavg;
	if(newTke)
		newTke[index] = tkeavg;
	if(newEpsilon)
		newEpsilon[index] = epsavg;
}


//! Identifies particles which form the free-surface
template<KernelType kerneltype, BoundaryType boundarytype, flag_t simflags, bool savenormals>
__global__ void
calcSurfaceparticleDevice(	const	float4*			posArray,
									float4*			normals,
									particleinfo*	newInfo,
							const	hashKey*		particleHash,
							const	uint*			cellStart,
							const	neibdata*		neibsList,
							const	uint			numParticles,
							const	float			slength,
							const	float			influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	particleinfo info = tex1Dfetch(infoTex, index);

	#if PREFER_L1
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 normal = make_float4(0.0f);

	if (NOT_FLUID(info) || INACTIVE(pos)) {
		if (savenormals)
			normals[index] = make_float4(NAN);
		return;
	}

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	CLEAR_FLAG(info, FG_SURFACE);

	// self contribution to normalization: W(0)*vol
	normal.w = W<kerneltype>(0.0f, slength)*pos.w/physical_density(tex1Dfetch(velTex, index).w,fluid_num(info));

	// First loop over all neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length3(relPos);

		// read neighbor data from sorted arrays
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// neighbor volume
		const float neib_vol = relPos.w/physical_density(tex1Dfetch(velTex, neib_index).w, fluid_num(neib_info));

		if (r < influenceradius) {
			const float f = F<kerneltype>(r, slength)*neib_vol; // 1/r ∂Wij/∂r Vj
			normal.x -= f * relPos.x;
			normal.y -= f * relPos.y;
			normal.z -= f * relPos.z;
			normal.w += W<kerneltype>(r, slength)*neib_vol;	// Wij*Vj ;

		}
	}

	// Checking the planes
	if (simflags & ENABLE_PLANES)
		for (uint i = 0; i < d_numplanes; ++i) {
			const float r = PlaneDistance(gridPos, as_float3(pos), d_plane[i]);
			if (r < influenceradius) {
				// since our current normal is still unnormalized, the plane normal
				// contribution must be scaled up to match the length of the current normal
				as_float3(normal) += d_plane[i].normal*length3(normal);
			}
		}

	const float normal_length = length3(normal);

	int nc = 0;

	// Second loop over all neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length3(relPos);

		float cosconeangle;

		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius) {
			float criteria = -dot3(normal, relPos);
			if (FLUID(neib_info))
				cosconeangle = d_cosconeanglefluid;
			else
				cosconeangle = d_cosconeanglenonfluid;

			if (criteria > r*normal_length*cosconeangle)
				nc++;
		}

	}

	if (!nc)
		SET_FLAG(info, FG_SURFACE);

	newInfo[index] = info;

	if (savenormals) {
		normal.x /= normal_length;
		normal.y /= normal_length;
		normal.z /= normal_length;
		normals[index] = normal;
		}

}

//! Identifies particles at the interface of two fluids and at the free-surface
template<KernelType kerneltype, BoundaryType boundarytype, flag_t simflags, bool savenormals>
__global__ void
calcInterfaceparticleDevice(	const	float4*			posArray,
									float4*			normals,
									particleinfo*	newInfo,
							const	hashKey*		particleHash,
							const	uint*			cellStart,
							const	neibdata*		neibsList,
							const	uint			numParticles,
							const	float			deltap,
							const	float			slength,
							const	float			influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	particleinfo info = tex1Dfetch(infoTex, index);

	#if PREFER_L1
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 normal_fs = make_float4(0.0f); // free-surface
	float4 normal_if = make_float4(0.0f); // interface

	if (NOT_FLUID(info) || INACTIVE(pos)) {
		// NOTE: inactive particles will keep their last surface flag status
		newInfo[index] = info;
		if (savenormals)
			normals[index] = make_float4(NAN);
		return;
	}

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	CLEAR_FLAG(info, FG_SURFACE);
	CLEAR_FLAG(info, FG_INTERFACE);

	// Particle physical density and volume
	const float p_rho = physical_density(tex1Dfetch(velTex, index).w,fluid_num(info));
	const float p_volume = pos.w/p_rho;

	// self contribution to normalization: W(0)*vol
	normal_fs.w = W<kerneltype>(0.0f, slength)*p_volume;
	normal_if.w = W<kerneltype>(0.0f, slength)*p_volume;

	// First loop over all neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
#if PREFER_L1
				posArray[neib_index]
#else
				tex1Dfetch(posTex, neib_index)
#endif
				);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;


		const particleinfo n_info = tex1Dfetch(infoTex, neib_index);
		const ParticleType nptype = PART_TYPE(n_info);
		const float r = length(as_float3(relPos));

		// neighbor physical density and volume
		const float n_rho = physical_density(tex1Dfetch(velTex, neib_index).w, fluid_num(n_info));
		const float n_volume = relPos.w/n_rho;

		if (r < influenceradius) {
			const float f = F<kerneltype>(r, slength); // 1/r ∂Wij/∂r
			normal_fs.x -= f * relPos.x;
			normal_fs.y -= f * relPos.y;
			normal_fs.z -= f * relPos.z;
			normal_fs.w += W<kerneltype>(r, slength)*n_volume;	// Wij*Vj ;
		}
		if (r < influenceradius && (fluid_num(info) == fluid_num(n_info) || NOT_FLUID(n_info))) {
			const float f = F<kerneltype>(r, slength); // 1/r ∂Wij/∂r
			normal_if.x -= f * relPos.x;
			normal_if.y -= f * relPos.y;
			normal_if.z -= f * relPos.z;
			normal_if.w += W<kerneltype>(r, slength)*n_volume;	// Wij*Vj ;
		}
	}

		normal_fs.x *= p_volume;
		normal_fs.y *= p_volume;
		normal_fs.z *= p_volume;

		normal_if.x *= p_volume;
		normal_if.y *= p_volume;
		normal_if.z *= p_volume;

	const float normal_fs_length = length3(normal_fs);
	const float normal_if_length = length3(normal_if);

	int nc_fs = 0;
	int nc_if = 0;

	// Second loop over all neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length3(relPos);

		float cosconeangle;

		const particleinfo n_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius) {
			float criteria_fs = -dot3(normal_fs, relPos);
			if (FLUID(n_info))
				cosconeangle = d_cosconeanglefluid;
			else
				cosconeangle = d_cosconeanglenonfluid;


			if (criteria_fs > r*normal_fs_length*cosconeangle)
				nc_fs++;
		}
		if (r < influenceradius && (fluid_num(info) == fluid_num(n_info) || NOT_FLUID(n_info))) {
			float criteria_if = -dot3(normal_if, relPos);
			if (FLUID(n_info))
				cosconeangle = d_cosconeanglefluid;
			else
				cosconeangle = d_cosconeanglenonfluid;

			//cosconeangle = d_cosconeanglefluid;

			if (criteria_if > r*normal_if_length*cosconeangle)
				nc_if++;
		}
	}

	if (!nc_fs)
		SET_FLAG(info, FG_SURFACE);

	if (!nc_if && nc_fs)
		SET_FLAG(info, FG_INTERFACE);

	newInfo[index] = info;

	if (savenormals) {
		normal_fs.x /= normal_fs_length;
		normal_fs.y /= normal_fs_length;
		normal_fs.z /= normal_fs_length;

		normal_if.x /= normal_if_length;
		normal_if.y /= normal_if_length;
		normal_if.z /= normal_if_length;

		if (!nc_if && nc_fs) {
			normals[index] = normal_if;
		} else {
			normals[index] = normal_fs;
		}
	}
}

//! Identifies particles at the interface of two fluids and at the free-surface
template<KernelType kerneltype, BoundaryType boundarytype, flag_t simflags, bool savenormals>
__global__ void
calcInterfaceparticleDevice(	const	float4*			posArray,
									float4*			normals,
									particleinfo*	newInfo,
							const	float2 *		vertPos0,
							const	float2 *		vertPos1,
							const	float2 *		vertPos2,
							const	hashKey*		particleHash,
							const	uint*			cellStart,
							const	neibdata*		neibsList,
							const	uint			numParticles,
							const	float			deltap,
							const	float			slength,
							const	float			influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	particleinfo info = tex1Dfetch(infoTex, index);

	#if PREFER_L1
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif
	float4 normal_fs = make_float4(0.0f); // free-surface
	float4 normal_if = make_float4(0.0f); // interface

	if (NOT_FLUID(info) || INACTIVE(pos)) {
		// NOTE: inactive particles will keep their last surface flag status
		newInfo[index] = info;
		if (savenormals)
			normals[index] = make_float4(NAN);
		return;
	}

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	CLEAR_FLAG(info, FG_SURFACE);
	CLEAR_FLAG(info, FG_INTERFACE);

	// Particle physical density and volume
	const float p_rho = physical_density(tex1Dfetch(velTex, index).w,fluid_num(info));
	const float p_volume = pos.w/p_rho;

	// self contribution to normalization: W(0)*vol
	normal_fs.w = W<kerneltype>(0.0f, slength)*p_volume;
	normal_if.w = W<kerneltype>(0.0f, slength)*p_volume;

	// First loop over all neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
#if PREFER_L1
				posArray[neib_index]
#else
				tex1Dfetch(posTex, neib_index)
#endif
				);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;


		const particleinfo n_info = tex1Dfetch(infoTex, neib_index);
		const ParticleType nptype = PART_TYPE(n_info);
		const float r = length(as_float3(relPos));

		// neighbor physical density and volume
		const float n_rho = physical_density(tex1Dfetch(velTex, neib_index).w, fluid_num(n_info));

		if (boundarytype == SA_BOUNDARY) {
			if (nptype == PT_FLUID || nptype == PT_VERTEX) {
				const float n_volume = relPos.w/n_rho;
				const float n_volume0 = relPos.w/d_rho0[fluid_num(n_info)];
				float n_theta = n_volume/n_volume0;

				if (r < influenceradius) {
					const float f = F<kerneltype>(r, slength); // 1/r ∂Wij/∂r
					normal_fs.x -= f * n_theta * relPos.x;
					normal_fs.y -= f * n_theta * relPos.y;
					normal_fs.z -= f * n_theta * relPos.z;
					normal_fs.w += W<kerneltype>(r, slength)*n_volume;	// Wij*Vj ;
				}
				if (r < influenceradius && (fluid_num(info) == fluid_num(n_info) || NOT_FLUID(n_info))) {
					const float f = F<kerneltype>(r, slength); // 1/r ∂Wij/∂r
					normal_if.x -= f * n_theta * relPos.x;
					normal_if.y -= f * n_theta * relPos.y;
					normal_if.z -= f * n_theta * relPos.z;
					normal_if.w += W<kerneltype>(r, slength)*n_volume;	// Wij*Vj ;
				}
			} else if (nptype == PT_BOUNDARY && r < influenceradius) {
				const float4 belem = tex1Dfetch(boundTex, neib_index);
				const float3 normal_s = as_float3(tex1Dfetch(boundTex, neib_index));
				const float3 q = as_float3(relPos)/slength;
				float3 q_vb[3];
				calcVertexRelPos(q_vb, belem,
						vertPos0[neib_index], vertPos1[neib_index], vertPos2[neib_index], slength);
				const float ggamAS = gradGamma<kerneltype>(slength, q, q_vb, normal_s);
				/* Actual volume should be calculated from the actual (interpolated) density
				 * and the mass. But boundary elements do not have mass. To get their actual 
				 * interpolated volume, we use the fact that:
				 *
				 * 	 n_ref_volume/n_ref_volume0 = n_rho0/n_rho
				 * 	 ==> n_ref_volume = n_ref_volume0*n_rho0/n_rho 
				 * 
				 * with n_ref_volume0 = deltap^3. Note that we are computing the reference volume
				 * here which is defined by:
				 *
				 * 	n_ref_volume = n_volume/n_theta
				 */
				const float n_ref_volume0 = deltap*deltap*deltap;
				const float n_ref_volume = n_ref_volume0*d_rho0[fluid_num(n_info)]/n_rho;


				// Free-surface
				normal_fs.x += ggamAS / n_ref_volume * relPos.x * normal_s.x;
				normal_fs.y += ggamAS / n_ref_volume * relPos.y * normal_s.y;
				normal_fs.z += ggamAS / n_ref_volume * relPos.z * normal_s.z;
				// Interface
				normal_if.x += ggamAS / n_ref_volume * relPos.x * normal_s.x;
				normal_if.y += ggamAS / n_ref_volume * relPos.y * normal_s.y;
				normal_if.z += ggamAS / n_ref_volume * relPos.z * normal_s.z;
			}
		} else {
			const float n_volume = relPos.w/n_rho;
			if (r < influenceradius) {
				const float f = F<kerneltype>(r, slength); // 1/r ∂Wij/∂r
				normal_fs.x -= f * relPos.x;
				normal_fs.y -= f * relPos.y;
				normal_fs.z -= f * relPos.z;
				normal_fs.w += W<kerneltype>(r, slength)*n_volume;	// Wij*Vj ;
			}
			if (r < influenceradius && (fluid_num(info) == fluid_num(n_info) || NOT_FLUID(n_info))) {
				const float f = F<kerneltype>(r, slength); // 1/r ∂Wij/∂r
				normal_if.x -= f * relPos.x;
				normal_if.y -= f * relPos.y;
				normal_if.z -= f * relPos.z;
				normal_if.w += W<kerneltype>(r, slength)*n_volume;	// Wij*Vj ;
			}
		}
	}

	if (boundarytype == SA_BOUNDARY) {
		const float gamma = tex1Dfetch(gamTex, index).w;
		normal_fs.x *= p_volume/gamma;
		normal_fs.y *= p_volume/gamma;
		normal_fs.z *= p_volume/gamma;
		normal_fs.w /= gamma;

		normal_if.x *= p_volume/gamma;
		normal_if.y *= p_volume/gamma;
		normal_if.z *= p_volume/gamma;
		normal_if.w /= gamma;
	} else {
		normal_fs.x *= p_volume;
		normal_fs.y *= p_volume;
		normal_fs.z *= p_volume;

		normal_if.x *= p_volume;
		normal_if.y *= p_volume;
		normal_if.z *= p_volume;
	}

	const float normal_fs_length = length3(normal_fs);
	const float normal_if_length = length3(normal_if);

	int nc_fs = 0;
	int nc_if = 0;

	// Second loop over all neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length3(relPos);

		float cosconeangle;

		const particleinfo n_info = tex1Dfetch(infoTex, neib_index);

		if (r < influenceradius) {
			float criteria_fs = -dot3(normal_fs, relPos);
			cosconeangle = d_cosconeanglefluid;

			if (criteria_fs > r*normal_fs_length*cosconeangle)
				nc_fs++;
		}
		if (r < influenceradius && (fluid_num(info) == fluid_num(n_info) || NOT_FLUID(n_info))) {
			float criteria_if = -dot3(normal_if, relPos);
			cosconeangle = d_cosconeanglefluid;

			if (criteria_if > r*normal_if_length*cosconeangle)
				nc_if++;
		}
	}

	if (!nc_fs)
		SET_FLAG(info, FG_SURFACE);

	if (!nc_if && nc_fs)
		SET_FLAG(info, FG_INTERFACE);

	newInfo[index] = info;

	if (savenormals) {
		normal_fs.x /= normal_fs_length;
		normal_fs.y /= normal_fs_length;
		normal_fs.z /= normal_fs_length;

		normal_if.x /= normal_if_length;
		normal_if.y /= normal_if_length;
		normal_if.z /= normal_if_length;

		if (!nc_if && nc_fs) {
			normals[index] = normal_if;
		} else {
			normals[index] = normal_fs;
		}
	}
}


// TODO documentation
__global__ void
fluxComputationDevice
			(	const	particleinfo	*pinfo,
				const	float4			*eulerVel,
				const	float4			*boundElement,
						float			*d_IOflux,
				const	uint			numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if(index < numParticles) {
		const particleinfo info = pinfo[index];
		if (IO_BOUNDARY(info) && BOUNDARY(info)) {
			const float4 normal = boundElement[index];
			atomicAdd(&d_IOflux[object(info)], normal.w*dot3(eulerVel[index],normal));
		}
	}
}

/************************************************************************************************************/
/*					   Parallel reduction kernels															*/
/************************************************************************************************************/

extern __shared__ float4 shmem4[];

//! Computes the energy of all particles
extern "C" __global__
void calcEnergiesDevice(
	const		float4	*pPos,
	const		float4	*pVel,
	const	particleinfo	*pInfo,
	const		hashKey	*particleHash,
		uint	numParticles,
		uint	numFluids,
		float4	*output
		)
{
	// shared memory for this kernel should be sized to
	// blockDim.x*numFluids*sizeof(float4)*2

	uint gid = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;
	uint stride = INTMUL(gridDim.x,blockDim.x);
	// .x kinetic, .y potential, .z internal
	float4 energy[MAX_FLUID_TYPES], E_k[MAX_FLUID_TYPES];

#pragma unroll
	for (uint i = 0; i < MAX_FLUID_TYPES; ++i)
		energy[i] = E_k[i] = make_float4(0.0f);

	while (gid < numParticles) {
		const float4 pos = pPos[gid];
		const float4 vel = pVel[gid];
		const int3 gridPos = calcGridPosFromParticleHash( particleHash[gid] );
		particleinfo pinfo = pInfo[gid];
		if (FLUID(pinfo)) {
			uint fnum = fluid_num(pinfo);
			float v2 = kahan_sqlength(as_float3(vel));
			// TODO improve precision by splitting the float part from the grid part
			float gh = kahan_dot(d_gravity, as_float3(pos) + (make_float3(gridPos) + 0.5f)*d_cellSize);
			kahan_add(energy[fnum].x, pos.w*v2/2, E_k[fnum].x);
			kahan_add(energy[fnum].y, -pos.w*gh, E_k[fnum].y);
			// internal elastic energy
			float gamma = d_gammacoeff[fnum];
			float gm1 = d_gammacoeff[fnum]-1;
			float rho0 = d_rho0[fnum];
			float elen = __powf(vel.w/rho0, gm1)/gm1 + rho0/vel.w - gamma/gm1;
			float ssp = soundSpeed(vel.w, fnum);
			elen *= ssp*ssp/gamma;
			kahan_add(energy[fnum].z, pos.w*elen, E_k[fnum].z);
		}
		gid += stride;
	}

	uint lid = threadIdx.x;
	for (uint offset = blockDim.x/2; offset; offset >>= 1) {
		stride = offset*numFluids; // stride between fields in shmem4 memory
		if (lid >= offset && lid < 2*offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = lid + offset*i;
				shmem4[idx] = energy[i];
				idx += stride;
				shmem4[idx] = E_k[i];
			}
		}
		__syncthreads();
		if (lid < offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = lid + offset*(i+1);
				float4 other = shmem4[idx];
				idx += stride;
				float4 oth_k = shmem4[idx];
				kahan_add(energy[i].x, oth_k.x, E_k[i].x);
				kahan_add(energy[i].x, other.x, E_k[i].x);
				kahan_add(energy[i].y, oth_k.y, E_k[i].y);
				kahan_add(energy[i].y, other.y, E_k[i].y);
				kahan_add(energy[i].z, oth_k.z, E_k[i].z);
				kahan_add(energy[i].z, other.z, E_k[i].z);
			}
		}
	}

	if (lid == 0) {
		for (uint i = 0; i < numFluids; ++i) {
			output[blockIdx.x + INTMUL(gridDim.x,i)] = energy[i];
			output[blockIdx.x + INTMUL(gridDim.x,numFluids+i)] = E_k[i];
		}
	}
}

//! Sum the previously computed energy up (across threads)
extern "C" __global__
void calcEnergies2Device(
		float4* buffer,
		uint	prev_blocks,
		uint	numFluids)
{
	// shared memory for this kernel should be sized to
	// blockDim.x*numFluids*sizeof(float4)*2

	uint gid = threadIdx.x;
	float4 energy[MAX_FLUID_TYPES];
	float4 E_k[MAX_FLUID_TYPES];
	for (uint i = 0; i < numFluids; ++i) {
		if (gid < prev_blocks) {
			energy[i] = buffer[gid + prev_blocks*i];
			E_k[i] = buffer[gid + prev_blocks*(numFluids+i)];
		} else {
			energy[i] = E_k[i] = make_float4(0.0f);
		}
	}

	uint stride;
	for (uint offset = blockDim.x/2; offset; offset >>= 1) {
		stride = offset*numFluids; // stride between fields in shmem4 memory
		if (gid >= offset && gid < 2*offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = gid + offset*i;
				shmem4[idx] = energy[i];
				idx += stride;
				shmem4[idx] = E_k[i];
			}
		}
		__syncthreads();
		if (gid < offset) {
			for (uint i = 0; i < numFluids; ++i) {
				uint idx = gid + offset*(i+1);
				float4 other = shmem4[idx];
				idx += stride;
				float4 oth_k = shmem4[idx];
				kahan_add(energy[i].x, oth_k.x, E_k[i].x);
				kahan_add(energy[i].x, other.x, E_k[i].x);
				kahan_add(energy[i].y, oth_k.y, E_k[i].y);
				kahan_add(energy[i].y, other.y, E_k[i].y);
				kahan_add(energy[i].z, oth_k.z, E_k[i].z);
				kahan_add(energy[i].z, other.z, E_k[i].z);
			}
		}
	}

	if (gid == 0) {
		for (uint i = 0; i < numFluids; ++i)
			buffer[i] = energy[i] + E_k[i];
	}
}

} //namespace cupostprocess

#endif
