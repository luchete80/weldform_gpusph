#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

/*
 * Device code.
 */

#ifndef _FORCES_KERNEL_
#define _FORCES_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "vector_math.h"
#include "multi_gpu_defines.h"
#include "GlobalData.h"

#include "kahan.h"
#include "tensor.cu"

#include "device_core.cu"

//#include "visc_kernel.cu"


#if __COMPUTE__ < 20
#define printf(...) /* eliminate printf from 1.x */
#endif

// Single-precision M_PI
// FIXME : ah, ah ! Single precision with 976896587958795795 decimals ....
#define M_PIf 3.141592653589793238462643383279502884197169399375105820974944f

#define MAXKASINDEX 10

/** \namespace cuforces
 *  \brief Contains all device functions/kernels/variables used force computations, filters and boundary conditions
 *
 *  The namespace cuforces contains all the device part of force computations, filters and boundary conditions :
 *  	- device constants/variables
 *  	- device functions
 *  	- kernels
 *
 *  \ingroup forces
 */
namespace cuforces {

using namespace cugeom;
using namespace cusph;
using namespace cuphys;
using namespace cuneibs;
//using namespace cuvisc;

// Core SPH functions
/** \name Device constants
 *  @{ */
// Rigid body data
__constant__ int3	d_rbcgGridPos[MAX_BODIES]; //< cell of the center of gravity
__constant__ float3	d_rbcgPos[MAX_BODIES]; //< in-cell coordinate of the center of gravity
__constant__ int	d_rbstartindex[MAX_BODIES];
/*  @} */

/** \name Device functions
 *  @{ */

/************************************************************************************************************/
/*							  Functions used by the different CUDA kernels							        */
/************************************************************************************************************/

//! Lennard-Jones boundary repulsion force
__device__ __forceinline__ float
LJForce(const float r)
{
	float force = 0.0f;

	if (r <= d_r0)
		force = d_dcoeff*(__powf(d_r0/r, d_p1coeff) - __powf(d_r0/r, d_p2coeff))/(r*r);

	return force;
}

//! Monaghan-Kajtar boundary repulsion force
/*!
 Monaghan-Kajtar boundary repulsion force doi:10.1016/j.cpc.2009.05.008
 to be multiplied by r_aj vector
 we allow the fluid particle mass mass_f to be different from the
 boundary particle mass mass_b even though they are typically the same
 (except for multi-phase fluids)
*/
__device__ __forceinline__ float
MKForce(const float r, const float slength,
		const float mass_f, const float mass_b)
{
	// MK always uses the 1D cubic or quintic Wendland spline
	float w = 0.0f;

	float force = 0.0f;

	// Wendland has radius 2
	if (r <= 2*slength) {	//TODO: fixme use influenceradius
		float qq = r/slength;
		w = 1.8f * __powf(1.0f - 0.5f*qq, 4.0f) * (2.0f*qq + 1.0f);  //TODO: optimize
		// float dist = r - d_MK_d;
		float dist = max(d_epsartvisc, r - d_MK_d);
		force = d_MK_K*w*2*mass_b/(d_MK_beta * dist * r * (mass_f+mass_b));
	}

	return force;
}
/************************************************************************************************************/

/******************** Functions for computing repulsive force directly from DEM *****************************/

// TODO: check for the maximum timestep

//! Computes normal and viscous force wrt to solid planar boundary
__device__ __forceinline__ float
PlaneForce(	const int3&		gridPos,
			const float3&	pos,
			const float		mass,
			const plane_t&	plane,
			const float3&	vel,
			const float		dynvisc,
			float4&			force)
{
	// relative position of our particle from the reference point of the plane
	const float r = PlaneDistance(gridPos, pos, plane);
	if (r < d_r0) {
		const float DvDt = LJForce(r);
		// Unitary normal vector of the surface
		const float3 relPos = plane.normal*r;

		as_float3(force) += DvDt*relPos;

		// tangential velocity component
		const float3 v_t = vel - dot(vel, relPos)/r*relPos/r; //TODO: check

		// f = -µ u/∆n

		// viscosity
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0-r*r)/(pos.w*r);
		// float coeff = -dynvisc*M_PI*(d_r0*d_r0*3/(M_PI*2)-r*r)/(pos.w*r);
		const float coeff = -dynvisc*d_partsurf/(mass*r);

		// coeff should not be higher than needed to nil v_t in the maximum allowed dt
		// coefficients are negative, so the smallest in absolute value is the biggest

		/*
		float fmag = length(as_float3(force));
		float coeff2 = -sqrt(fmag/slength)/(d_dtadaptfactor*d_dtadaptfactor);
		if (coeff2 < -d_epsartvisc)
			coeff = max(coeff, coeff2);
			*/

		as_float3(force) += coeff*v_t;

		return -coeff;
	}

	return 0.0f;
}

//! DOC-TODO Describe function
__device__ __forceinline__ float
GeometryForce(	const int3&		gridPos,
				const float3&	pos,
				const float		mass,
				const float3&	vel,
				const float		dynvisc,
				float4&			force)
{
	float coeff_max = 0.0f;
	for (uint i = 0; i < d_numplanes; ++i) {
		float coeff = PlaneForce(gridPos, pos, mass, d_plane[i], vel, dynvisc, force);
		if (coeff > coeff_max)
			coeff_max = coeff;
	}

	return coeff_max;
}

//! DOC-TODO describe function
__device__ __forceinline__ float
DemLJForce(	const texture<float, 2, hipReadModeElementType> texref,
			const int3&	gridPos,
			const float3&	pos,
			const float		mass,
			const float3&	vel,
			const float		dynvisc,
			float4&			force)
{
	const float2 demPos = DemPos(gridPos, pos);

	const float globalZ = d_worldOrigin.z + (gridPos.z + 0.5f)*d_cellSize.z + pos.z;
	const float globalZ0 = DemInterpol(texref, demPos);

	if (globalZ - globalZ0 < d_demzmin) {
		const plane_t demPlane(DemTangentPlane(texref, gridPos, pos, demPos, globalZ0));

		return PlaneForce(gridPos, pos, mass, demPlane, vel, dynvisc, force);
	}
	return 0;
}

/************************************************************************************************************/

/************************************************************************************************************/
/*		Device functions used in kernels other than the main forces kernel									*/
/************************************************************************************************************/

//! contribution of neighbor at relative position relPos with weight w to the MLS matrix mls
__device__ __forceinline__ void
MlsMatrixContrib(symtensor4 &mls, float4 const& relPos, float w)
{
	mls.xx += w;						// xx = ∑Wij*Vj
	mls.xy += relPos.x*w;				// xy = ∑(xi - xj)*Wij*Vj
	mls.xz += relPos.y*w;				// xz = ∑(yi - yj)*Wij*Vj
	mls.xw += relPos.z*w;				// xw = ∑(zi - zj)*Wij*Vj
	mls.yy += relPos.x*relPos.x*w;		// yy = ∑(xi - xj)^2*Wij*Vj
	mls.yz += relPos.x*relPos.y*w;		// yz = ∑(xi - xj)(yi - yj)*Wij*Vj
	mls.yw += relPos.x*relPos.z*w;		// yz = ∑(xi - xj)(zi - zj)*Wij*Vj
	mls.zz += relPos.y*relPos.y*w;		// zz = ∑(yi - yj)^2*Wij*Vj
	mls.zw += relPos.y*relPos.z*w;		// zz = ∑(yi - yj)(zi - zj)*Wij*Vj
	mls.ww += relPos.z*relPos.z*w;		// zz = ∑(yi - yj)^2*Wij*Vj

}

//! MLS contribution
/*!
 contribution of neighbor at relative position relPos with weight w to the
 MLS correction when B is the first row of the inverse MLS matrix
*/
__device__ __forceinline__ float
MlsCorrContrib(float4 const& B, float4 const& relPos, float w)
{
	return (B.x + B.y*relPos.x + B.z*relPos.y + B.w*relPos.z)*w;
	// ρ = ∑(ß0 + ß1(xi - xj) + ß2(yi - yj))*Wij*Vj
}

/*  @} */

/** \name Kernels
 *  @{ */

/************************************************************************************************************/

/************************************************************************************************************/
/*										Density computation							*/
/************************************************************************************************************/

//! Continuity equation with the Grenier formulation
/*!
 When using the Grenier formulation, density is reinitialized at each timestep from
 a Shepard-corrected mass distribution limited to same-fluid particles M and volumes ω computed
 from a continuity equation, with ρ = M/ω.
 During the same run, we also compute σ, the discrete specific volume
 (see e.g. Hu & Adams 2005), obtained by summing the kernel computed over
 _all_ neighbors (not just the same-fluid ones) which is used in the continuity
 equation as well as the Navier-Stokes equation
*/
template<KernelType kerneltype, BoundaryType boundarytype>
__global__ void
densityGrenierDevice(
			float* __restrict__		sigmaArray,
	const	float4* __restrict__	posArray,
			float4* __restrict__	velArray,
	const	particleinfo* __restrict__	infoArray,
	const	hashKey* __restrict__	particleHash,
	const	float4* __restrict__	volArray,
	const	uint* __restrict__		cellStart,
	const	neibdata* __restrict__	neibsList,
	const	uint	numParticles,
	const	float	slength,
	const	float	influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = infoArray[index];

	/* We only process FLUID particles normally,
	   except with DYN_BOUNDARY, where we also process boundary particles
	   */
	if (boundarytype != DYN_BOUNDARY && NOT_FLUID(info))
		return;

	const float4 pos = posArray[index];

	if (INACTIVE(pos))
		return;

	const ushort fnum = fluid_num(info);
	const float vol = volArray[index].w;
	float4 vel = velArray[index];

	// self contribution
	float corr = W<kerneltype>(0, slength);
	float sigma = corr;
	float mass_corr = pos.w*corr;

	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// For DYN_BOUNDARY particles, we compute sigma in the same way as fluid particles,
	// except that if the boundary particle has no fluid neighbors we set its
	// sigma to a default value which is the 'typical' specific volume, given by
	// the typical number of neighbors divided by the volume of the influence sphere
	bool has_fluid_neibs = false;

	// Loop over all FLUID neighbors, and over BOUNDARY neighbors if using
	// DYN_BOUNDARY
	// TODO: check with SA
	for_each_neib2(PT_FLUID, (boundarytype == DYN_BOUNDARY ? PT_BOUNDARY : PT_NONE),
			index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		const particleinfo neib_info = infoArray[neib_index];
		float r = length(as_float3(relPos));

		/* Contributions only come from active particles within the influence radius
		   that are fluid particles (or also non-fluid in DYN_BOUNDARY case).
		   Sigma calculations uses all such particles, whereas smoothed mass
		   only uses same-fluid particles.
		   Note that this requires PT_BOUNDARY neighbors to be in the list for
		   PT_BOUNDARY particles, lest the boundary particles end up assuming
		   they are always on the free surface.
		   TODO an alternative approach for DYN_BOUNDARY would be to assign
		   the sigma from the closest fluid particle, but that would require
		   two runs, one for fluid and one for neighbor particles.
		 */
		if (INACTIVE(relPos) || r >= influenceradius)
			continue;

		const float w = W<kerneltype>(r, slength);
		sigma += w;
		if (FLUID(neib_info))
			has_fluid_neibs = true;

		/* For smoothed mass, fluid particles only consider fluid particles,
		   and non-fluid (only present for DYN_BOUNDARY) only consider non-fluid
		   */
		if ((boundarytype != DYN_BOUNDARY || (PART_TYPE(neib_info) == PART_TYPE(info)))
			&& fluid_num(neib_info) == fnum) {
			mass_corr += relPos.w*w;
			corr += w;
		}
	}

	if (boundarytype == DYN_BOUNDARY && NOT_FLUID(info) && !has_fluid_neibs) {
		// TODO OPTIMIZE
		const float typical_sigma = 3*(cuneibs::d_maxFluidBoundaryNeibs)/
			(4*M_PIf*influenceradius*influenceradius*influenceradius);
		sigma = typical_sigma;
	}

	// M = mass_corr/corr, ρ = M/ω
	// this could be optimized to pos.w/vol assuming all same-fluid particles
	// have the same mass
	vel.w = mass_corr/(corr*vol);
	vel.w = numerical_density(vel.w,fnum);
	velArray[index] = vel;
	sigmaArray[index] = sigma;
}

/************************************************************************************************************/


/************************************************************************************************************/
/*					   Kernels for computing acceleration without gradient correction					 */
/************************************************************************************************************/

/* forcesDevice kernel and auxiliary types and functions */
#include "forces_kernel.def"

/************************************************************************************************************/


/************************************************************************************************************/
/*					   Kernels for XSPH, Shepard and MLS corrections									   */
/************************************************************************************************************/

//! This kernel computes the Sheppard correction
template<KernelType kerneltype,
	BoundaryType boundarytype>
__global__ void
__launch_bounds__(BLOCK_SIZE_SHEPARD, MIN_BLOCKS_SHEPARD)
shepardDevice(	const float4*	posArray,
				float4*			newVel,
				const hashKey*		particleHash,
				const uint*		cellStart,
				const neibdata*	neibsList,
				const uint		numParticles,
				const float		slength,
				const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = tex1Dfetch(infoTex, index);

	#if PREFER_L1
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// If particle is inactive there is absolutely nothing to do
	if (INACTIVE(pos))
		return;

	float4 vel = tex1Dfetch(velTex, index);

	// We apply Shepard normalization :
	//	* with LJ or DYN boundary only on fluid particles
	//TODO 	* with SA boundary ???
	// in any other case we have to copy the vel vector in the new velocity array
	if (NOT_FLUID(info)) {
		newVel[index] = vel;
		return;
	}


	// Taking into account self contribution in summation
	float temp1 = pos.w*W<kerneltype>(0, slength);
	float temp2 = temp1/physical_density(vel.w,fluid_num(info)) ;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Loop over all FLUID neighbors, and over BOUNDARY neighbors if using
	// DYN_BOUNDARY
	// TODO: check with SA
	for_each_neib2(PT_FLUID, (boundarytype == DYN_BOUNDARY ? PT_BOUNDARY : PT_NONE),
			index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Skip inactive neighbors
		if (INACTIVE(relPos))
			continue;

		const float r = length(as_float3(relPos));

		const float neib_rho = physical_density(tex1Dfetch(velTex, neib_index).w,fluid_num(neib_info));

		if (r < influenceradius ) {
			const float w = W<kerneltype>(r, slength)*relPos.w;
			temp1 += w;
			temp2 += w/neib_rho;
		}
	}

	// Normalize the density and write in global memory
	vel.w = numerical_density(temp1/temp2,fluid_num(info));
	newVel[index] = vel;
}

//! This kernel computes the MLS correction
template<KernelType kerneltype,
	BoundaryType boundarytype>
__global__ void
__launch_bounds__(BLOCK_SIZE_MLS, MIN_BLOCKS_MLS)
MlsDevice(	const float4*	posArray,
			float4*			newVel,
			const hashKey*		particleHash,
			const uint*		cellStart,
			const neibdata*	neibsList,
			const uint		numParticles,
			const float		slength,
			const float		influenceradius)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const particleinfo info = tex1Dfetch(infoTex, index);

	#if PREFER_L1
	const float4 pos = posArray[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// If particle is inactive there is absolutely nothing to do
	if (INACTIVE(pos))
		return;

	float4 vel = tex1Dfetch(velTex, index);

	// We apply MLS normalization :
	//	* with LJ or DYN boundary only on fluid particles
	//TODO 	* with SA boundary ???
	// in any other case we have to copy the vel vector in the new velocity array
	//if (NOT_FLUID(info)) {
	//	newVel[index] = vel;
	//	return;
	//}

	// MLS matrix elements
	symtensor4 mls;
	mls.xx = mls.xy = mls.xz = mls.xw =
		mls.yy = mls.yz = mls.yw =
		mls.zz = mls.zw = mls.ww = 0;

	// Number of neighbors
	int neibs_num = 0;

	// Taking into account self contribution in MLS matrix construction
	mls.xx = W<kerneltype>(0, slength)*pos.w/physical_density(vel.w,fluid_num(info));

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// First loop over neighbors
	// Loop over all FLUID neighbors, and over BOUNDARY neighbors if using
	// DYN_BOUNDARY
	// TODO: check with SA
	for_each_neib2(PT_FLUID, (boundarytype == DYN_BOUNDARY ? PT_BOUNDARY : PT_NONE),
			index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// Skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length(as_float3(relPos));
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);
		const float neib_rho = physical_density(tex1Dfetch(velTex, neib_index).w,fluid_num(neib_info));
 

		// Add neib contribution only if it's a fluid one
		if (r < influenceradius) {
			neibs_num ++;
			const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;	// Wij*Vj

			/* Scale relPos by slength for stability and resolution independence */
			MlsMatrixContrib(mls, relPos/slength, w);
		}
	} // end of first loop trough neighbors

	// We want to compute B solution of M B = E where E =(1, 0, 0, 0) and
	// M is our MLS matrix. M is symmetric, positive (semi)definite. Since we
	// cannot guarantee that the matrix is invertible (it won't be in cases
	// such as thin sheets of particles or structures of even lower topological
	// dimension), we rely on the iterative conjugate residual method to
	// find a solution, with E itself as initial guess.

	// known term
	const float4 E = make_float4(1, 0, 0, 0);

	const float D = det(mls);

	// solution
	float4 B;
	if (fabsf(D) < FLT_EPSILON) {
		symtensor4 mls_eps = mls;
		const float eps = fabsf(D) + FLT_EPSILON;
		mls_eps.xx += eps;
		mls_eps.yy += eps;
		mls_eps.zz += eps;
		mls_eps.ww += eps;
		const float D_eps = det(mls_eps);
		B = adjugate_row1(mls_eps)/D_eps;
	} else {
		B = adjugate_row1(mls)/D;
	}

#define MAX_CR_STEPS 32
	uint steps = 0;
	for (; steps < MAX_CR_STEPS; ++steps) {
		float lenB = hypot(B);

		float4 MdotB = dot(mls, B);
		float4 residual = E - MdotB;

		// r.M.r
		float num = ddot(mls, residual);

		// (M.r).(M.r)
		float4 Mp = dot(mls, residual);
		float den = dot(Mp, Mp);

		float4 corr = (num/den)*residual;
		float lencorr = hypot(corr);

		if (hypot(residual) < lenB*FLT_EPSILON)
			break;

		if (lencorr < 2*lenB*FLT_EPSILON)
			break;

		B += corr;
	}

	/* Scale for resolution independence, again */
	B.y /= slength;
	B.z /= slength;
	B.w /= slength;

	// Taking into account self contribution in density summation
	vel.w = B.x*W<kerneltype>(0, slength)*pos.w;

	// Second loop over neighbors
	// Loop over all FLUID neighbors, and over BOUNDARY neighbors if using
	// DYN_BOUNDARY
	// TODO: check with SA
	for_each_neib2(PT_FLUID, (boundarytype == DYN_BOUNDARY ? PT_BOUNDARY : PT_NONE),
			index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(
		#if PREFER_L1
			posArray[neib_index]
		#else
			tex1Dfetch(posTex, neib_index)
		#endif
			);

		// Skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length(as_float3(relPos));

		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Interaction between two particles
		if (r < influenceradius && (boundarytype == DYN_BOUNDARY || FLUID(neib_info))) {
			const float w = W<kerneltype>(r, slength)*relPos.w;	 // ρj*Wij*Vj = mj*Wij
			vel.w += MlsCorrContrib(B, relPos, w);
		}


	}  // end of second loop trough neighbors

	// If MLS starts misbehaving, define DEBUG_PARTICLE: this will
	// print the MLS-corrected density for the particles statisfying
	// the DEBUG_PARTICLE condition. Some examples:

//#define DEBUG_PARTICLE (index == numParticles - 1)
//#define DEBUG_PARTICLE (id(info) == numParticles - 1)
//#define DEBUG_PARTICLE (fabs(err) > 64*FLT_EPSILON)

#ifdef DEBUG_PARTICLE
	{
		const float old = tex1Dfetch(velTex, index).w;
		const float err = 1 - vel.w/old;
		if (DEBUG_PARTICLE) {
			printf("MLS %d %d %22.16g => %22.16g (%6.2e)\n",
				index, id(info),
				old, vel.w, err*100);
		}
	}
#endif
        vel.w = numerical_density(vel.w,fluid_num(info));
	newVel[index] = vel;
}
/************************************************************************************************************/

/************************************************************************************************************/
/*					   CFL max kernel																		*/
/************************************************************************************************************/
//! Computes the max of an array of floats
/** 
 * Each thread reads 4 elements at a time, computing the max of these four elements (hence why
 * the input type is float4 and not float).
 * The launch grid “slides” over the entire input array, which is compused by numquarts float4s.
 * Each block reduces the per-thread reductions in shared memory, and then writes out a single float.
 */
template <unsigned int blockSize>
__global__ void
fmaxDevice(
	float * __restrict__ output, //< output array,
	const float4 * __restrict__ input, //< input array
	const uint numquarts)
{
	__shared__ float sdata[blockSize];

	/* Step #1: reduction from global memory into a private register */

	// Size of the sliding window
	const unsigned int stride = blockSize*gridDim.x;

	unsigned int i = blockIdx.x*blockSize + threadIdx.x;

	// Accumulator
	float myMax = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < numquarts)
	{
		float4 in = input[i];
		myMax = fmaxf(myMax, fmaxf(
				fmaxf(in.x, in.y),
				fmaxf(in.z, in.w)));
		i += stride;
	}

	// each thread puts its local sum into shared memory
	const unsigned int tid = threadIdx.x;

	sdata[tid] = myMax;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = myMax = fmaxf(myMax,sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = myMax = fmaxf(myMax,sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = myMax = fmaxf(myMax,sdata[tid +  64]); } __syncthreads(); }

	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and induce incorrect behavior.
	if (tid < 32)
	{
		volatile float* smem = sdata;
		if (blockSize >=  64) { smem[tid] = myMax = fmaxf(myMax, smem[tid + 32]); }
		if (blockSize >=  32) { smem[tid] = myMax = fmaxf(myMax, smem[tid + 16]); }
		if (blockSize >=  16) { smem[tid] = myMax = fmaxf(myMax, smem[tid +  8]); }
		if (blockSize >=   8) { smem[tid] = myMax = fmaxf(myMax, smem[tid +  4]); }
		if (blockSize >=   4) { smem[tid] = myMax = fmaxf(myMax, smem[tid +  2]); }
		if (blockSize >=   2) { smem[tid] = myMax = fmaxf(myMax, smem[tid +  1]); }
	}

	// write result for this block to global mem
	if (tid == 0)
		output[blockIdx.x] = myMax;
}
/************************************************************************************************************/

/** @} */

/************************************************************************************************************/

} //namespace cuforces
#endif
