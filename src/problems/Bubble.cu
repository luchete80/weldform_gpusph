#include "hip/hip_runtime.h"
/*  Copyright (c) 2015-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */


#include <iostream>

#include "Bubble.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define USE_PLANES 0

#if USE_PLANES
#define USE_GHOST 1 // set to 0 for standard planes
#else
#define USE_GHOST 0 // never use ghosts unless using planes
#endif



Bubble::Bubble(GlobalData *_gdata) : Problem(_gdata),
	dyn_layers(0)
{
	// Size and origin of the simulation domain
	R = 0.025;
	H = 10*R;
	lx = ly = 6*R;
	lz = H;

	// TODO GAUSSIAN kernel of radius 3
	SETUP_FRAMEWORK(
		formulation<SPH_GRENIER>,
		//formulation<SPH_F2>,
		viscosity<DYNAMICVISC>,
		boundary<DYN_BOUNDARY>,
		add_flags<ENABLE_MULTIFLUID | (USE_PLANES ? ENABLE_PLANES : ENABLE_NONE) |
              ENABLE_REPACKING>
	);

	// SPH parameters
	// Grenier sets h/R = 0.128
	//set_deltap(6.72e-4/1.3);
	set_deltap(0.128*R/1.3);

	if (simparams()->boundarytype == DYN_BOUNDARY) {
		dyn_layers = simparams()->get_influence_layers() + 1;
		extra_offset = make_double3(dyn_layers*m_deltap);
	} else {
		dyn_layers = 0;
		extra_offset = make_double3(0.0);
	}
	m_size = make_double3(lx, ly, lz) + 2*extra_offset;
	m_origin = -m_size/2;

	simparams()->buildneibsfreq = 10;

	simparams()->tend = 1.0;

	set_interface_epsilon(0.08);

	// Physical parameters
	set_gravity(-9.81f);
	float g = get_gravity_magnitude();

	setMaxFall(H);

	float maxvel = sqrt(g*H);
	float rho0 = 1;
	float rho1 = 1000;
	float c0_air = 198*maxvel;
	float c0_water = 14*maxvel;

	air = add_fluid(rho0);
	water = add_fluid(rho1);

	set_equation_of_state(air,  1.4, c0_air);
	set_equation_of_state(water,  7.0f, c0_water);

	// Repacking options
	simparams()->repack_maxiter = 1000;
	simparams()->repack_a = 100/(2.*c0_air*c0_air);
	simparams()->repack_alpha = 2*m_deltap/c0_air;

	set_kinematic_visc(air, 4.5e-3f);
	set_kinematic_visc(water, 3.5e-5f);

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Name of problem used for directory creation
	m_name = "Bubble";

	setPositioning(PP_CORNER);
	GeometryID experiment_box = addBox(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(m_origin),
		m_size.x, m_size.y, m_size.z);
	disableCollisions(experiment_box);

	GeometryID fluid = addBox(GT_FLUID, FT_SOLID,
		Point(m_origin + extra_offset),
		lx, ly, H);

	// the actual particle mass will be set during the
	// initializeParticles routine, both for the tank and for the
	// fluid particles, by multiplitying the mass computed here
	// by the density of the particle
	setParticleMassByDensity(experiment_box, 1);
	setParticleMassByDensity(fluid, 1);

}

void Bubble::copy_planes(PlaneList &planes)
{
#if USE_PLANES
	// z = m_origin.z
	planes.push_back( implicit_plane(0, 0, 1.0, -m_origin.z) );
	// z = m_origin.z+lz
	planes.push_back( implicit_plane(0, 0, -1.0, m_origin.z+lz) );
	// y = m_origin.y
	planes.push_back( implicit_plane(0, 1.0, 0, -m_origin.y) );
	// y = m_origin.y+ly
	planes.push_back( implicit_plane(0, -1.0, 0, m_origin.y+ly) );
	// x = m_origin.x
	planes.push_back( implicit_plane(1.0, 0, 0, -m_origin.x) );
	// x = m_origin.x+lx
	planes.push_back( implicit_plane(-1.0, 0, 0, m_origin.x+lx) );
#endif
}


// the bubble is initially located centered at 2R from the bottom.
bool is_inside(double3 const& origin, float R, double4 const& pt)
{
	return
		(pt.x*pt.x) +
		(pt.y*pt.y) +
		(pt.z - (origin.z+2*R))*(pt.z - (origin.z+2*R)) < R*R;
}

// Mass and density initialization
void
Bubble::initializeParticles(BufferList &buffers, const uint numParticles)
{
	// Example usage

	// 1. warn the user if this is expected to take much time
	printf("Initializing particles density and mass...\n");

	// 2. grab the particle arrays from the buffer list
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	double4 *pos_global = buffers.getData<BUFFER_POS_GLOBAL>();
	float4 *pos = buffers.getData<BUFFER_POS>();

	// 3. iterate on the particles
	for (uint i = 0; i < numParticles; i++) {
		float rho = 1;
		double depth = H - pos_global[i].z + m_origin.z;
		// for boundary particles, we use the density of water,
		// fluid particles will override fluid_idx depending on whether
		// they are inside the bubble or not
		int fluid_idx = water;
		if (FLUID(info[i])) {
			fluid_idx = is_inside(m_origin, R, pos_global[i]) ? air : water;
			// hydrostatic density: for the heavy fluid, this is simply computed
			// as the density that gives pressure rho g h, with h depth
			rho = hydrostatic_density(depth, fluid_idx);
			// for the bubble, the hydrostatic density must be computed in a slightly
			// more complex way:
			if (fluid_idx == air) {
				// interface: depth of center of the bubble corrected by
				// R^2 - horizontal offset squared
				// note: no correction by m_origin.z because we are only
				// interested in deltas
				float z_intf = 2*R + sqrtf(R*R
						- (pos_global[i].x)*(pos_global[i].x)
						- (pos_global[i].y)*(pos_global[i].y)
						);
				// pressure at interface, from heavy fluid
				float g = get_gravity_magnitude();
				float P = physparams()->rho0[water]*(H - z_intf)*g;
				// plus hydrostatic pressure from _our_ fluid
				P += physparams()->rho0[air]*(z_intf - pos_global[i].z + m_origin.z)*g;
				rho = density_for_pressure(P, air);
			}
			info[i]= make_particleinfo(PT_FLUID, fluid_idx, i);
		} else if (BOUNDARY(info[i])) {
			rho = hydrostatic_density(depth, fluid_idx);
			info[i]= make_particleinfo(PT_BOUNDARY, fluid_idx, i);
		}
		// fix up the particle mass according to the actual density
		pos[i].w *= physical_density(rho, fluid_idx);
		vel[i].w = rho;
	}
}


bool Bubble::need_write(double t) const
{
 	return 0;
}




