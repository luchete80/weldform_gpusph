#include "hip/hip_runtime.h"
/*  Copyright (c) 2015-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <string>
#include <iostream>

#include "Spheric2SA.h"
#include "GlobalData.h"
#include "cudasimframework.cu"
#include "textures.cuh"
#include "utils.h"

#define USE_PLANES 0

Spheric2SA::Spheric2SA(GlobalData *_gdata) : Problem(_gdata)
{
	SETUP_FRAMEWORK(
		viscosity<KEPSVISC>,
		//viscosity<DYNAMICVISC>,
		boundary<SA_BOUNDARY>,
		periodicity<PERIODIC_NONE>,
		kernel<WENDLAND>,
		densitydiffusion<FERRARI>,
		add_flags<ENABLE_GAMMA_QUADRATURE>
	);

	set_deltap(0.02715f);

	size_t water = add_fluid(1000.0);
	set_equation_of_state(water,  7.0f, 130.f);
	set_kinematic_visc(water, 1.0e-6f);
	set_gravity(-9.81f);

	simparams()->tend = 1.2;
	addPostProcess(SURFACE_DETECTION);
	addPostProcess(TESTPOINTS);
	H = 0.55;
	l = 3.5+0.02; w = 1.0+0.02; h = 2.0;
	m_origin = make_double3(-0.01, -0.01, -0.01);
	simparams()->ferrariLengthScale = 0.161f;
	resize_neiblist(128+128, 64);
	setMaxFall(H);

	// SPH parameters
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 1;
	simparams()->nlexpansionfactor = 1.1;

	// Size and origin of the simulation domain
	m_size = make_double3(l, w ,h);

	// Physical parameters
	float g = get_gravity_magnitude();

	// Drawing and saving times
	add_writer(VTKWRITER, 1e-2f);

	// Name of problem used for directory creation
	m_name = "Spheric2SA";

	// Building the geometry
	addHDF5File(GT_FLUID, Point(0,0,0), "./data_files/Spheric2/0.spheric2.fluid.h5sph", NULL);

	GeometryID container =
		addHDF5File(GT_FIXED_BOUNDARY, Point(0,0,0), "./data_files/Spheric2/0.spheric2.boundary.kent0.h5sph", NULL);
	disableCollisions(container);

	// Add water level gages
	add_gage(m_origin + make_double3(2.724, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));
	add_gage(m_origin + make_double3(2.228, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));
	add_gage(m_origin + make_double3(1.732, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));
	add_gage(m_origin + make_double3(0.582, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));

	// Pressure probes
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.021) + make_double3(0.01, 0.01, 0.01)); // the (0.01,0.01,0.01) vector accounts for the slightly shifted origin
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.061) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.101) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.141) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.4165, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.4565, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.4965, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.5365, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));

}

void
Spheric2SA::initializeParticles(BufferList &buffers, const uint numParticles)
{
	printf("k and epsilon initialization...\n");

	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	double4 *pos = buffers.getData<BUFFER_POS_GLOBAL>();
	float *k = buffers.getData<BUFFER_TKE>();
	float *epsilon = buffers.getData<BUFFER_EPSILON>();

	for (uint i = 0; i < numParticles; i++) {
		const float Ti = 0.01f;
		const float u = 1.0f; // TODO set according to initial velocity
		const float L = 1.0f; // TODO set according to geometry
		if (k && epsilon) {
			k[i] = fmaxf(1e-5f, 3.0f/2.0f*(u*Ti)*(u*Ti));
			epsilon[i] = fmaxf(1e-5f, 2.874944542f*k[i]*u*Ti/L);
			//k[i] = k0;
			//e[i] = 1.0f/0.41f/fmax(1.0f-fabs(z),0.5f*(float)m_deltap);
		}
	}
}

uint
Spheric2SA::max_parts(uint numpart)
{
	// gives an estimate for the maximum number of particles
	return numpart;
}

void Spheric2SA::fillDeviceMap()
{
	fillDeviceMapByAxis(X_AXIS);
}

bool Spheric2SA::need_write(double t) const
{
 	return 0;
}




