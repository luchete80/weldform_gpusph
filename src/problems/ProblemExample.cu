#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "ProblemExample.h"
/*#include "Cube.h"
#include "Point.h"
#include "Vector.h"
#include "GlobalData.h"*/
#include "cudasimframework.cu"

ProblemExample::ProblemExample(GlobalData *_gdata) : Problem(_gdata)
{
	SETUP_FRAMEWORK(
		// TODO update from legacy viscous models
		// laminar viscosity type: KINEMATICVISC, DYNAMICVISC
		// turbulent viscosity type: ARTVISC, SPSVISC, KEPSVISC
		viscosity<ARTVISC>,
		// boundary types: LJ_BOUNDARY, MK_BOUNDARY, SA_BOUNDARY, DYN_BOUNDARY
		boundary<LJ_BOUNDARY>,
		add_flags<ENABLE_PLANES>
	);

	// *** Initialization of minimal physical parameters
	set_deltap(0.02f);
	set_gravity(-9.81);
	setMaxFall(3.0);
	float g = get_gravity_magnitude();
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.0f);
	//set_kinematic_visc(0, 1.0e-2f);

	// *** Initialization of minimal simulation parameters
	resize_neiblist(256, 32);

	// *** Other parameters and settings
	add_writer(VTKWRITER, 1e-1f);
	m_name = "ProblemExample";

	// *** Post-processing
	// In our case we show an example of how to add the problem-specific
	// CALC_PRIVATE post-processing. Additional post-processing functions
	// are defined in PostProcessType
	addPostProcess(CALC_PRIVATE);

	// domain size
	const double dimX = 10;
	const double dimY = 10;
	const double dimZ = 3;

	// world size
	m_origin = make_double3(0, 0, 0);
	// NOTE: NAN value means that will be computed automatically
	m_size = make_double3(dimX, dimY, dimZ);

	// size and height of grid of cubes
	const double cube_size = 0.4;
	const double cube_Z = 1;

	// size and height of spheres of water
	const double sphere_radius = 0.5;
	const double sphere_Z = 2;

	// will create a grid of cubes and spheres
	const double grid_size = dimX / 5;
	const uint cubes_grid_size = 4;
	const uint spheres_grid_size = 3;

	// every geometry will be centered in the given coordinate
	setPositioning(PP_CENTER);

	// create infinite floor
	addPlane(0, 0, 1, 0);

	// origin of the grid of cubes and spheres
	const double cornerXY = (dimX / 2) - (grid_size / 2);

	// grid of cubes
	for (uint i=0; i < cubes_grid_size; i++)
		for (uint j=0; j < cubes_grid_size; j++) {
			// create cube
			GeometryID current = addCube(GT_FIXED_BOUNDARY, FT_BORDER,
				Point( cornerXY + i*grid_size/(cubes_grid_size-1),
				cornerXY + j*grid_size/(cubes_grid_size-1), cube_Z), cube_size);
			// rotate it
			rotate(current, i * (M_PI/2) / cubes_grid_size, j * (M_PI/2) / cubes_grid_size, 0);
		}

	// grid of spheres
	for (uint i=0; i < spheres_grid_size; i++)
		for (uint j=0; j < spheres_grid_size; j++)
			addSphere(GT_FLUID, FT_SOLID,
				Point( cornerXY + i*grid_size/(spheres_grid_size-1),
				cornerXY + j*grid_size/(spheres_grid_size-1), sphere_Z), sphere_radius);

	// setMassByDensity(floating_obj, physparams()->rho0[0] / 2);
}

using namespace hipcubounds; // to access calcGridPosFromParticleHash in device code
using namespace cuneibs; // to access iterators over neighbors

//! Compute a private variable
/*!
 This function computes an arbitrary passive array. It can be used for
 debugging purposes or passive scalars.

 In this example we simply compute the number of neighbors.
 */
template<BoundaryType boundarytype>
__global__ void
calcPrivateDevice(
			float*		priv,
	const	float4*		__restrict__	posArray,
	const	float4*		__restrict__		velArray,
	const	particleinfo* __restrict__	infoArray,
	const	hashKey*	__restrict__	particleHash,
	const	uint*		__restrict__	cellStart,
	const	neibdata*	__restrict__	neibsList,
	const	float		slength,
	const	float		inflRadius,
			uint		numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	const float4 pos = posArray[index];
	// To access the particle info and e.g. filter action based on particle type:
	//const particleinfo info = infoArray[index];
	// To access the particle velocity and density, e.g. to apply the standard SPH smoothing
	//const float4 vel = velArray[index];

	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	uint neibs = 0;

	// Loop over all the neighbors
	for_every_neib(boundarytype, index, pos, gridPos, cellStart, neibsList) {

		const uint neib_index = neib_iter.neib_index();

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		const float4 relPos = neib_iter.relPos(posArray[neib_index]);

		float r = length3(relPos);
		if (r < inflRadius)
			neibs += 1;
	}

	// Will convert to float on storage, because BUFFER_PRIVATE is a float buffer
	priv[index] = neibs;
}

void ProblemExample::calcPrivate(flag_t options,
	BufferList const& bufread,
	BufferList & bufwrite,
	uint numParticles,
	uint particleRangeEnd,
	uint deviceIndex,
	const GlobalData * const gdata)
{
	/* Example of typical implementation */

	// thread per particle
	uint numThreads = BLOCK_SIZE_CALCTEST;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float4 *pos = bufread.getData<BUFFER_POS>();
	const float4 *vel = bufread.getData<BUFFER_VEL>();
	const particleinfo *info = bufread.getData<BUFFER_INFO>();
	const hashKey *particleHash = bufread.getData<BUFFER_HASH>();
	const uint *cellStart = bufread.getData<BUFFER_CELLSTART>();
	const neibdata *neibsList = bufread.getData<BUFFER_NEIBSLIST>();

	float *priv = bufwrite.getData<BUFFER_PRIVATE>();

	//execute kernel
	calcPrivateDevice<LJ_BOUNDARY><<<numBlocks, numThreads>>>(
			priv,
			pos, vel, info,
			particleHash,
			cellStart,
			neibsList,
			simparams()->slength,
			simparams()->influenceRadius,
			numParticles);

	KERNEL_CHECK_ERROR;
}

std::string
ProblemExample::get_private_name(flag_t buffer) const
{
	return "NeibsNum";
}
