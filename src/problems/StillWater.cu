#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "StillWater.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define CENTER_DOMAIN 0
// set to coords (x,y,z) if more accuracy is needed in such point
// (waiting for relative coordinates)
#if CENTER_DOMAIN
#define OFFSET_X (-l/2)
#define OFFSET_Y (-w/2)
#define OFFSET_Z (-h/2)
#else
#define OFFSET_X 0
#define OFFSET_Y 0
#define OFFSET_Z 0
#endif

StillWater::StillWater(GlobalData *_gdata) : Problem(_gdata)
{
	m_usePlanes = get_option("use-planes", false); // --use-planes true to enable use of planes for boundaries
	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 16); // --ppH N to change deltap to H/N

	// density diffusion terms, see DensityDiffusionType
	const DensityDiffusionType rhodiff = get_option("density-diffusion", FERRARI);

	SETUP_FRAMEWORK(
		//viscosity<KINEMATICVISC>,
		viscosity<DYNAMICVISC>,
		//viscosity<ARTVISC>,
		boundary<DYN_BOUNDARY>
		//boundary<LJ_BOUNDARY>
	).select_options(
		rhodiff,
		m_usePlanes, add_flags<ENABLE_PLANES>()
	);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	H = 1;

	set_deltap(H/ppH);

	setMaxFall(H);

	l = w = sqrt(2)*H; h = 1.1*H;

	// Size and origin of the simulation domain
	m_size = make_double3(l, w ,h);
	m_origin = make_double3(OFFSET_X, OFFSET_Y, OFFSET_Z);

	// SPH parameters
	set_timestep(0.00004f);
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 20;
	simparams()->ferrariLengthScale = H;

	// enlarge the domain to take into account the extra layers of particles
	// of the boundary
	if (simparams()->boundarytype == DYN_BOUNDARY && !m_usePlanes) {
		// number of layers
		dyn_layers = ceil(simparams()->kernelradius*simparams()->sfactor);
		// extra layers are one less (since other boundary types still have
		// one layer)
		double3 extra_offset = make_double3((dyn_layers-1)*m_deltap);
		m_origin -= extra_offset;
		m_size += 2*extra_offset;
	} else {
		dyn_layers = 1;
	}

	simparams()->tend = 100.0;
	if (simparams()->boundarytype == SA_BOUNDARY) {
		resize_neiblist(128, 128);
	};

	// Physical parameters
	set_gravity(-9.81f);
	const float g = get_gravity_magnitude();
	const float maxvel = sqrt(2*g*H);
	// purely for cosmetic reason, let's round the soundspeed to the next
	// integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0, 7.0f, c0);

	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	//set_kinematic_visc(0, 1.0e-6f);

	// Drawing and saving times
	add_writer(VTKWRITER, 1.0);

	// Name of problem used for directory creation
	m_name = "StillWater";

	// Building the geometry
	setPositioning(PP_CORNER);
	// distance between fluid box and wall
	float wd = m_deltap;

	GeometryID experiment_box = addBox(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(m_origin), m_size.x, m_size.y, m_size.z);
	disableCollisions(experiment_box);

	m_fluidOrigin = m_origin;
	if (dyn_layers > 1) // shift by the extra offset of the experiment box
		m_fluidOrigin += make_double3((dyn_layers)*m_deltap);
	m_fluidOrigin += make_double3(wd); // one wd space from the boundary
	double shift = 2*wd;
	if (dyn_layers > 1)
		shift = (dyn_layers-1)*m_deltap*2;
	GeometryID fluid = addBox(GT_FLUID, FT_SOLID,
		m_fluidOrigin, l-shift, w-shift, H-shift);

}

void StillWater::copy_planes(PlaneList& planes)
{
	if (!m_usePlanes) return;

	planes.push_back( implicit_plane(0, 0, 1.0, -m_origin.z) );
	planes.push_back( implicit_plane(0, 1.0, 0, -m_origin.x) );
	planes.push_back( implicit_plane(0, -1.0, 0, m_origin.x + w) );
	planes.push_back( implicit_plane(1.0, 0, 0, -m_origin.y) );
	planes.push_back( implicit_plane(-1.0, 0, 0, m_origin.y + l) );
}

